#include "hip/hip_runtime.h"
/*
   Implements the sequential cuda vectors.
*/

#define PETSC_SKIP_SPINLOCK

#include <petscconf.h>
#include <petsc/private/vecimpl.h>
#include <../src/vec/vec/impls/dvecimpl.h>
#include <../src/vec/vec/impls/seq/seqcuda/cudavecimpl.h>

#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

/*
    Allocates space for the vector array on the GPU if it does not exist.
    Does NOT change the PetscCUDAFlag for the vector
    Does NOT zero the CUDA array

 */
PetscErrorCode VecCUDAAllocateCheck(Vec v)
{
  PetscErrorCode ierr;
  hipError_t    err;
  hipStream_t   stream;
  Vec_CUDA       *veccuda;

  PetscFunctionBegin;
  if (!v->spptr) {
    ierr = PetscMalloc(sizeof(Vec_CUDA),&v->spptr);CHKERRQ(ierr);
    veccuda = (Vec_CUDA*)v->spptr;
    err = hipMalloc((void**)&veccuda->GPUarray_allocated,sizeof(PetscScalar)*((PetscBLASInt)v->map->n));CHKERRCUDA(err);
    veccuda->GPUarray = veccuda->GPUarray_allocated;
    err = hipStreamCreate(&stream);CHKERRCUDA(err);
    veccuda->stream = stream;
    veccuda->hostDataRegisteredAsPageLocked = PETSC_FALSE;
    if (v->valid_GPU_array == PETSC_OFFLOAD_UNALLOCATED) {
      if (v->data && ((Vec_Seq*)v->data)->array) {
        v->valid_GPU_array = PETSC_OFFLOAD_CPU;
      } else {
        v->valid_GPU_array = PETSC_OFFLOAD_GPU;
      }
    }
  }
  PetscFunctionReturn(0);
}

/* Copies a vector from the CPU to the GPU unless we already have an up-to-date copy on the GPU */
PetscErrorCode VecCUDACopyToGPU(Vec v)
{
  PetscErrorCode ierr;
  hipError_t    err;
  Vec_CUDA       *veccuda;
  PetscScalar    *varray;

  PetscFunctionBegin;
  PetscCheckTypeNames(v,VECSEQCUDA,VECMPICUDA);
  ierr = VecCUDAAllocateCheck(v);CHKERRQ(ierr);
  if (v->valid_GPU_array == PETSC_OFFLOAD_CPU) {
    ierr = PetscLogEventBegin(VEC_CUDACopyToGPU,v,0,0,0);CHKERRQ(ierr);
    veccuda=(Vec_CUDA*)v->spptr;
    varray=veccuda->GPUarray;
    err = hipMemcpy(varray,((Vec_Seq*)v->data)->array,v->map->n*sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);
    ierr = PetscLogEventEnd(VEC_CUDACopyToGPU,v,0,0,0);CHKERRQ(ierr);
    v->valid_GPU_array = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecCUDACopyToGPUSome(Vec v, PetscCUDAIndices ci)
{
  PetscScalar    *varray;
  PetscErrorCode ierr;
  hipError_t    err;
  PetscScalar    *cpuPtr, *gpuPtr;
  Vec_Seq        *s;
  VecScatterCUDAIndices_PtoP ptop_scatter = (VecScatterCUDAIndices_PtoP)ci->scatter;

  PetscFunctionBegin;
  PetscCheckTypeNames(v,VECSEQCUDA,VECMPICUDA);
  ierr = VecCUDAAllocateCheck(v);CHKERRQ(ierr);
  if (v->valid_GPU_array == PETSC_OFFLOAD_CPU) {
    s = (Vec_Seq*)v->data;

    ierr   = PetscLogEventBegin(VEC_CUDACopyToGPUSome,v,0,0,0);CHKERRQ(ierr);
    varray = ((Vec_CUDA*)v->spptr)->GPUarray;
    gpuPtr = varray + ptop_scatter->recvLowestIndex;
    cpuPtr = s->array + ptop_scatter->recvLowestIndex;

    /* Note : this code copies the smallest contiguous chunk of data
       containing ALL of the indices */
    err = hipMemcpy(gpuPtr,cpuPtr,ptop_scatter->nr*sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);

    // Set the buffer states
    v->valid_GPU_array = PETSC_OFFLOAD_BOTH;
    ierr = PetscLogEventEnd(VEC_CUDACopyToGPUSome,v,0,0,0);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}


/*
     VecCUDACopyFromGPU - Copies a vector from the GPU to the CPU unless we already have an up-to-date copy on the CPU
*/
PetscErrorCode VecCUDACopyFromGPU(Vec v)
{
  PetscErrorCode ierr;
  hipError_t    err;
  Vec_CUDA       *veccuda;
  PetscScalar    *varray;

  PetscFunctionBegin;
  PetscCheckTypeNames(v,VECSEQCUDA,VECMPICUDA);
  ierr = VecCUDAAllocateCheckHost(v);CHKERRQ(ierr);
  if (v->valid_GPU_array == PETSC_OFFLOAD_GPU) {
    ierr = PetscLogEventBegin(VEC_CUDACopyFromGPU,v,0,0,0);CHKERRQ(ierr);
    veccuda=(Vec_CUDA*)v->spptr;
    varray=veccuda->GPUarray;
    err = hipMemcpy(((Vec_Seq*)v->data)->array,varray,v->map->n*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(err);
    ierr = PetscLogEventEnd(VEC_CUDACopyFromGPU,v,0,0,0);CHKERRQ(ierr);
    v->valid_GPU_array = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

/* Note that this function only copies *some* of the values up from the GPU to CPU,
   which means that we need recombine the data at some point before using any of the standard functions.
   We could add another few flag-types to keep track of this, or treat things like VecGetArray VecRestoreArray
   where you have to always call in pairs
*/
PetscErrorCode VecCUDACopyFromGPUSome(Vec v, PetscCUDAIndices ci)
{
  const PetscScalar *varray, *gpuPtr;
  PetscErrorCode    ierr;
  hipError_t       err;
  PetscScalar       *cpuPtr;
  Vec_Seq           *s;
  VecScatterCUDAIndices_PtoP ptop_scatter = (VecScatterCUDAIndices_PtoP)ci->scatter;

  PetscFunctionBegin;
  PetscCheckTypeNames(v,VECSEQCUDA,VECMPICUDA);
  ierr = VecCUDAAllocateCheckHost(v);CHKERRQ(ierr);
  if (v->valid_GPU_array == PETSC_OFFLOAD_GPU) {
    ierr   = PetscLogEventBegin(VEC_CUDACopyFromGPUSome,v,0,0,0);CHKERRQ(ierr);

    varray=((Vec_CUDA*)v->spptr)->GPUarray;
    s = (Vec_Seq*)v->data;
    gpuPtr = varray + ptop_scatter->sendLowestIndex;
    cpuPtr = s->array + ptop_scatter->sendLowestIndex;

    /* Note : this code copies the smallest contiguous chunk of data
       containing ALL of the indices */
    err = hipMemcpy(cpuPtr,gpuPtr,ptop_scatter->ns*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(err);

    ierr = VecCUDARestoreArrayRead(v,&varray);CHKERRQ(ierr);
    ierr = PetscLogEventEnd(VEC_CUDACopyFromGPUSome,v,0,0,0);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/*MC
   VECSEQCUDA - VECSEQCUDA = "seqcuda" - The basic sequential vector, modified to use CUDA

   Options Database Keys:
. -vec_type seqcuda - sets the vector type to VECSEQCUDA during a call to VecSetFromOptions()

  Level: beginner

.seealso: VecCreate(), VecSetType(), VecSetFromOptions(), VecCreateSeqWithArray(), VECMPI, VecType, VecCreateMPI(), VecCreateSeq()
M*/

PetscErrorCode VecAYPX_SeqCUDA(Vec yin,PetscScalar alpha,Vec xin)
{
  const PetscScalar *xarray;
  PetscScalar       *yarray;
  PetscErrorCode    ierr;
  PetscBLASInt      one=1,bn;
  PetscScalar       sone=1.0;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    cberr;
  hipError_t       err;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(yin->map->n,&bn);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
  if (alpha == (PetscScalar)0.0) {
    err = hipMemcpy(yarray,xarray,bn*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(err);
  } else if (alpha == (PetscScalar)1.0) {
    cberr = cublasXaxpy(cublasv2handle,bn,&alpha,xarray,one,yarray,one);CHKERRCUBLAS(cberr);
    ierr = PetscLogFlops(2.0*yin->map->n);CHKERRQ(ierr);
  } else {
    cberr = cublasXscal(cublasv2handle,bn,&alpha,yarray,one);CHKERRCUBLAS(cberr);
    cberr = cublasXaxpy(cublasv2handle,bn,&sone,xarray,one,yarray,one);CHKERRCUBLAS(cberr);
    ierr = PetscLogFlops(2.0*yin->map->n);CHKERRQ(ierr);
  }
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecAXPY_SeqCUDA(Vec yin,PetscScalar alpha,Vec xin)
{
  const PetscScalar *xarray;
  PetscScalar       *yarray;
  PetscErrorCode    ierr;
  PetscBLASInt      one=1,bn;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    cberr;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  if (alpha != (PetscScalar)0.0) {
    ierr = PetscBLASIntCast(yin->map->n,&bn);CHKERRQ(ierr);
    ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDAGetArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
    cberr = cublasXaxpy(cublasv2handle,bn,&alpha,xarray,one,yarray,one);CHKERRCUBLAS(cberr);
    ierr = WaitForGPU();CHKERRCUDA(ierr);
    ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDARestoreArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
    ierr = PetscLogFlops(2.0*yin->map->n);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecPointwiseDivide_SeqCUDA(Vec win, Vec xin, Vec yin)
{
  PetscInt                              n = xin->map->n;
  const PetscScalar                     *xarray=NULL,*yarray=NULL;
  PetscScalar                           *warray=NULL;
  thrust::device_ptr<const PetscScalar> xptr,yptr;
  thrust::device_ptr<PetscScalar>       wptr;
  PetscErrorCode                        ierr;

  PetscFunctionBegin;
  ierr = VecCUDAGetArrayWrite(win,&warray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(yin,&yarray);CHKERRQ(ierr);
  try {
    wptr = thrust::device_pointer_cast(warray);
    xptr = thrust::device_pointer_cast(xarray);
    yptr = thrust::device_pointer_cast(yarray);
    thrust::transform(xptr,xptr+n,yptr,wptr,thrust::divides<PetscScalar>());
    ierr = WaitForGPU();CHKERRCUDA(ierr);
  } catch (char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Thrust error: %s", ex);
  }
  ierr = PetscLogFlops(n);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(yin,&yarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayWrite(win,&warray);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecWAXPY_SeqCUDA(Vec win,PetscScalar alpha,Vec xin, Vec yin)
{
  const PetscScalar *xarray=NULL,*yarray=NULL;
  PetscScalar       *warray=NULL;
  PetscErrorCode    ierr;
  PetscBLASInt      one=1,bn;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    cberr;
  hipError_t       err;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(win->map->n,&bn);CHKERRQ(ierr);
  if (alpha == (PetscScalar)0.0) {
    ierr = VecCopy_SeqCUDA(yin,win);CHKERRQ(ierr);
  } else {
    ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDAGetArrayRead(yin,&yarray);CHKERRQ(ierr);
    ierr = VecCUDAGetArrayWrite(win,&warray);CHKERRQ(ierr);
    err = hipMemcpy(warray,yarray,win->map->n*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(err);
    cberr = cublasXaxpy(cublasv2handle,bn,&alpha,xarray,one,warray,one);CHKERRCUBLAS(cberr);
    ierr = PetscLogFlops(2*win->map->n);CHKERRQ(ierr);
    ierr = WaitForGPU();CHKERRCUDA(ierr);
    ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDARestoreArrayRead(yin,&yarray);CHKERRQ(ierr);
    ierr = VecCUDARestoreArrayWrite(win,&warray);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecMAXPY_SeqCUDA(Vec xin, PetscInt nv,const PetscScalar *alpha,Vec *y)
{
  PetscErrorCode ierr;
  PetscInt       n = xin->map->n,j,j_rem;
  PetscScalar    alpha0,alpha1,alpha2,alpha3;

  PetscFunctionBegin;
  ierr = PetscLogFlops(nv*2.0*n);CHKERRQ(ierr);
  switch (j_rem=nv&0x3) {
    case 3:
      alpha0 = alpha[0];
      alpha1 = alpha[1];
      alpha2 = alpha[2];
      alpha += 3;
      ierr   = VecAXPY_SeqCUDA(xin,alpha0,y[0]);CHKERRQ(ierr);
      ierr   = VecAXPY_SeqCUDA(xin,alpha1,y[1]);CHKERRQ(ierr);
      ierr   = VecAXPY_SeqCUDA(xin,alpha2,y[2]);CHKERRQ(ierr);
      y   += 3;
      break;
    case 2:
      alpha0 = alpha[0];
      alpha1 = alpha[1];
      alpha +=2;
      ierr   = VecAXPY_SeqCUDA(xin,alpha0,y[0]);CHKERRQ(ierr);
      ierr   = VecAXPY_SeqCUDA(xin,alpha1,y[1]);CHKERRQ(ierr);
      y +=2;
      break;
    case 1:
      alpha0 = *alpha++;
      ierr   = VecAXPY_SeqCUDA(xin,alpha0,y[0]);CHKERRQ(ierr);
      y     +=1;
      break;
  }
  for (j=j_rem; j<nv; j+=4) {
    alpha0 = alpha[0];
    alpha1 = alpha[1];
    alpha2 = alpha[2];
    alpha3 = alpha[3];
    alpha += 4;
    ierr   = VecAXPY_SeqCUDA(xin,alpha0,y[0]);CHKERRQ(ierr);
    ierr   = VecAXPY_SeqCUDA(xin,alpha1,y[1]);CHKERRQ(ierr);
    ierr   = VecAXPY_SeqCUDA(xin,alpha2,y[2]);CHKERRQ(ierr);
    ierr   = VecAXPY_SeqCUDA(xin,alpha3,y[3]);CHKERRQ(ierr);
    y   += 4;
  }
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecDot_SeqCUDA(Vec xin,Vec yin,PetscScalar *z)
{
  const PetscScalar *xarray,*yarray;
  PetscErrorCode    ierr;
  PetscBLASInt      one=1,bn;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    cberr;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(yin->map->n,&bn);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(yin,&yarray);CHKERRQ(ierr);
  /* arguments y, x are reversed because BLAS complex conjugates the first argument, PETSc the second */
  cberr = cublasXdot(cublasv2handle,bn,yarray,one,xarray,one,z);CHKERRCUBLAS(cberr);
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  if (xin->map->n >0) {
    ierr = PetscLogFlops(2.0*xin->map->n-1);CHKERRQ(ierr);
  }
  ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(yin,&yarray);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

//
// CUDA kernels for MDot to follow
//

// set work group size to be a power of 2 (128 is usually a good compromise between portability and speed)
#define MDOT_WORKGROUP_SIZE 128
#define MDOT_WORKGROUP_NUM  128

#if !defined(PETSC_USE_COMPLEX)
// M = 2:
__global__ void VecMDot_SeqCUDA_kernel2(const PetscScalar *x,const PetscScalar *y0,const PetscScalar *y1,
                                        PetscInt size, PetscScalar *group_results)
{
  __shared__ PetscScalar tmp_buffer[2*MDOT_WORKGROUP_SIZE];
  PetscInt entries_per_group = (size - 1) / gridDim.x + 1;
  entries_per_group = (entries_per_group == 0) ? 1 : entries_per_group;  // for very small vectors, a group should still do some work
  PetscInt vec_start_index = blockIdx.x * entries_per_group;
  PetscInt vec_stop_index  = PetscMin((blockIdx.x + 1) * entries_per_group, size); // don't go beyond vec size

  PetscScalar entry_x    = 0;
  PetscScalar group_sum0 = 0;
  PetscScalar group_sum1 = 0;
  for (PetscInt i = vec_start_index + threadIdx.x; i < vec_stop_index; i += blockDim.x) {
    entry_x     = x[i];   // load only once from global memory!
    group_sum0 += entry_x * y0[i];
    group_sum1 += entry_x * y1[i];
  }
  tmp_buffer[threadIdx.x]                       = group_sum0;
  tmp_buffer[threadIdx.x + MDOT_WORKGROUP_SIZE] = group_sum1;

  // parallel reduction
  for (PetscInt stride = blockDim.x/2; stride > 0; stride /= 2) {
    __syncthreads();
    if (threadIdx.x < stride) {
      tmp_buffer[threadIdx.x                      ] += tmp_buffer[threadIdx.x+stride                      ];
      tmp_buffer[threadIdx.x + MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + MDOT_WORKGROUP_SIZE];
    }
  }

  // write result of group to group_results
  if (threadIdx.x == 0) {
    group_results[blockIdx.x]             = tmp_buffer[0];
    group_results[blockIdx.x + gridDim.x] = tmp_buffer[MDOT_WORKGROUP_SIZE];
  }
}

// M = 3:
__global__ void VecMDot_SeqCUDA_kernel3(const PetscScalar *x,const PetscScalar *y0,const PetscScalar *y1,const PetscScalar *y2,
                                        PetscInt size, PetscScalar *group_results)
{
  __shared__ PetscScalar tmp_buffer[3*MDOT_WORKGROUP_SIZE];
  PetscInt entries_per_group = (size - 1) / gridDim.x + 1;
  entries_per_group = (entries_per_group == 0) ? 1 : entries_per_group;  // for very small vectors, a group should still do some work
  PetscInt vec_start_index = blockIdx.x * entries_per_group;
  PetscInt vec_stop_index  = PetscMin((blockIdx.x + 1) * entries_per_group, size); // don't go beyond vec size

  PetscScalar entry_x    = 0;
  PetscScalar group_sum0 = 0;
  PetscScalar group_sum1 = 0;
  PetscScalar group_sum2 = 0;
  for (PetscInt i = vec_start_index + threadIdx.x; i < vec_stop_index; i += blockDim.x) {
    entry_x     = x[i];   // load only once from global memory!
    group_sum0 += entry_x * y0[i];
    group_sum1 += entry_x * y1[i];
    group_sum2 += entry_x * y2[i];
  }
  tmp_buffer[threadIdx.x]                           = group_sum0;
  tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] = group_sum1;
  tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] = group_sum2;

  // parallel reduction
  for (PetscInt stride = blockDim.x/2; stride > 0; stride /= 2) {
    __syncthreads();
    if (threadIdx.x < stride) {
      tmp_buffer[threadIdx.x                          ] += tmp_buffer[threadIdx.x+stride                          ];
      tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride +     MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 2 * MDOT_WORKGROUP_SIZE];
    }
  }

  // write result of group to group_results
  if (threadIdx.x == 0) {
    group_results[blockIdx.x                ] = tmp_buffer[0];
    group_results[blockIdx.x +     gridDim.x] = tmp_buffer[    MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 2 * gridDim.x] = tmp_buffer[2 * MDOT_WORKGROUP_SIZE];
  }
}

// M = 4:
__global__ void VecMDot_SeqCUDA_kernel4(const PetscScalar *x,const PetscScalar *y0,const PetscScalar *y1,const PetscScalar *y2,const PetscScalar *y3,
                                        PetscInt size, PetscScalar *group_results)
{
  __shared__ PetscScalar tmp_buffer[4*MDOT_WORKGROUP_SIZE];
  PetscInt entries_per_group = (size - 1) / gridDim.x + 1;
  entries_per_group = (entries_per_group == 0) ? 1 : entries_per_group;  // for very small vectors, a group should still do some work
  PetscInt vec_start_index = blockIdx.x * entries_per_group;
  PetscInt vec_stop_index  = PetscMin((blockIdx.x + 1) * entries_per_group, size); // don't go beyond vec size

  PetscScalar entry_x    = 0;
  PetscScalar group_sum0 = 0;
  PetscScalar group_sum1 = 0;
  PetscScalar group_sum2 = 0;
  PetscScalar group_sum3 = 0;
  for (PetscInt i = vec_start_index + threadIdx.x; i < vec_stop_index; i += blockDim.x) {
    entry_x     = x[i];   // load only once from global memory!
    group_sum0 += entry_x * y0[i];
    group_sum1 += entry_x * y1[i];
    group_sum2 += entry_x * y2[i];
    group_sum3 += entry_x * y3[i];
  }
  tmp_buffer[threadIdx.x]                           = group_sum0;
  tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] = group_sum1;
  tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] = group_sum2;
  tmp_buffer[threadIdx.x + 3 * MDOT_WORKGROUP_SIZE] = group_sum3;

  // parallel reduction
  for (PetscInt stride = blockDim.x/2; stride > 0; stride /= 2) {
    __syncthreads();
    if (threadIdx.x < stride) {
      tmp_buffer[threadIdx.x                          ] += tmp_buffer[threadIdx.x+stride                          ];
      tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride +     MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 2 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 3 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 3 * MDOT_WORKGROUP_SIZE];
    }
  }

  // write result of group to group_results
  if (threadIdx.x == 0) {
    group_results[blockIdx.x                ] = tmp_buffer[0];
    group_results[blockIdx.x +     gridDim.x] = tmp_buffer[    MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 2 * gridDim.x] = tmp_buffer[2 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 3 * gridDim.x] = tmp_buffer[3 * MDOT_WORKGROUP_SIZE];
  }
}

// M = 8:
__global__ void VecMDot_SeqCUDA_kernel8(const PetscScalar *x,const PetscScalar *y0,const PetscScalar *y1,const PetscScalar *y2,const PetscScalar *y3,
                                          const PetscScalar *y4,const PetscScalar *y5,const PetscScalar *y6,const PetscScalar *y7,
                                          PetscInt size, PetscScalar *group_results)
{
  __shared__ PetscScalar tmp_buffer[8*MDOT_WORKGROUP_SIZE];
  PetscInt entries_per_group = (size - 1) / gridDim.x + 1;
  entries_per_group = (entries_per_group == 0) ? 1 : entries_per_group;  // for very small vectors, a group should still do some work
  PetscInt vec_start_index = blockIdx.x * entries_per_group;
  PetscInt vec_stop_index  = PetscMin((blockIdx.x + 1) * entries_per_group, size); // don't go beyond vec size

  PetscScalar entry_x    = 0;
  PetscScalar group_sum0 = 0;
  PetscScalar group_sum1 = 0;
  PetscScalar group_sum2 = 0;
  PetscScalar group_sum3 = 0;
  PetscScalar group_sum4 = 0;
  PetscScalar group_sum5 = 0;
  PetscScalar group_sum6 = 0;
  PetscScalar group_sum7 = 0;
  for (PetscInt i = vec_start_index + threadIdx.x; i < vec_stop_index; i += blockDim.x) {
    entry_x     = x[i];   // load only once from global memory!
    group_sum0 += entry_x * y0[i];
    group_sum1 += entry_x * y1[i];
    group_sum2 += entry_x * y2[i];
    group_sum3 += entry_x * y3[i];
    group_sum4 += entry_x * y4[i];
    group_sum5 += entry_x * y5[i];
    group_sum6 += entry_x * y6[i];
    group_sum7 += entry_x * y7[i];
  }
  tmp_buffer[threadIdx.x]                           = group_sum0;
  tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] = group_sum1;
  tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] = group_sum2;
  tmp_buffer[threadIdx.x + 3 * MDOT_WORKGROUP_SIZE] = group_sum3;
  tmp_buffer[threadIdx.x + 4 * MDOT_WORKGROUP_SIZE] = group_sum4;
  tmp_buffer[threadIdx.x + 5 * MDOT_WORKGROUP_SIZE] = group_sum5;
  tmp_buffer[threadIdx.x + 6 * MDOT_WORKGROUP_SIZE] = group_sum6;
  tmp_buffer[threadIdx.x + 7 * MDOT_WORKGROUP_SIZE] = group_sum7;

  // parallel reduction
  for (PetscInt stride = blockDim.x/2; stride > 0; stride /= 2) {
    __syncthreads();
    if (threadIdx.x < stride) {
      tmp_buffer[threadIdx.x                          ] += tmp_buffer[threadIdx.x+stride                          ];
      tmp_buffer[threadIdx.x +     MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride +     MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 2 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 2 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 3 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 3 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 4 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 4 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 5 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 5 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 6 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 6 * MDOT_WORKGROUP_SIZE];
      tmp_buffer[threadIdx.x + 7 * MDOT_WORKGROUP_SIZE] += tmp_buffer[threadIdx.x+stride + 7 * MDOT_WORKGROUP_SIZE];
    }
  }

  // write result of group to group_results
  if (threadIdx.x == 0) {
    group_results[blockIdx.x                ] = tmp_buffer[0];
    group_results[blockIdx.x +     gridDim.x] = tmp_buffer[    MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 2 * gridDim.x] = tmp_buffer[2 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 3 * gridDim.x] = tmp_buffer[3 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 4 * gridDim.x] = tmp_buffer[4 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 5 * gridDim.x] = tmp_buffer[5 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 6 * gridDim.x] = tmp_buffer[6 * MDOT_WORKGROUP_SIZE];
    group_results[blockIdx.x + 7 * gridDim.x] = tmp_buffer[7 * MDOT_WORKGROUP_SIZE];
  }
}
#endif /* !defined(PETSC_USE_COMPLEX) */

PetscErrorCode VecMDot_SeqCUDA(Vec xin,PetscInt nv,const Vec yin[],PetscScalar *z)
{
  PetscErrorCode    ierr;
  PetscInt          i,n = xin->map->n,current_y_index = 0;
  const PetscScalar *xptr,*y0ptr,*y1ptr,*y2ptr,*y3ptr,*y4ptr,*y5ptr,*y6ptr,*y7ptr;
  PetscScalar       *group_results_gpu;
#if !defined(PETSC_USE_COMPLEX)
  PetscInt          j;
  PetscScalar       group_results_cpu[MDOT_WORKGROUP_NUM * 8]; // we process at most eight vectors in one kernel
#endif
  hipError_t    cuda_ierr;
  PetscBLASInt   one=1,bn;
  hipblasHandle_t cublasv2handle;
  hipblasStatus_t cberr;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(xin->map->n,&bn);CHKERRQ(ierr);
  if (nv <= 0) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"Number of vectors provided to VecMDot_SeqCUDA not positive.");
  /* Handle the case of local size zero first */
  if (!xin->map->n) {
    for (i=0; i<nv; ++i) z[i] = 0;
    PetscFunctionReturn(0);
  }

  // allocate scratchpad memory for the results of individual work groups:
  cuda_ierr = hipMalloc((void**)&group_results_gpu, sizeof(PetscScalar) * MDOT_WORKGROUP_NUM * 8);CHKERRCUDA(cuda_ierr);

  ierr = VecCUDAGetArrayRead(xin,&xptr);CHKERRQ(ierr);

  while (current_y_index < nv)
  {
    switch (nv - current_y_index) {

      case 7:
      case 6:
      case 5:
      case 4:
        ierr = VecCUDAGetArrayRead(yin[current_y_index  ],&y0ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+1],&y1ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+2],&y2ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+3],&y3ptr);CHKERRQ(ierr);

#if defined(PETSC_USE_COMPLEX)
        cberr = cublasXdot(cublasv2handle,bn,y0ptr,one,xptr,one,&z[current_y_index]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y1ptr,one,xptr,one,&z[current_y_index+1]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y2ptr,one,xptr,one,&z[current_y_index+2]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y3ptr,one,xptr,one,&z[current_y_index+3]);CHKERRCUBLAS(cberr);
#else
        // run kernel:
        VecMDot_SeqCUDA_kernel4<<<MDOT_WORKGROUP_NUM,MDOT_WORKGROUP_SIZE>>>(xptr,y0ptr,y1ptr,y2ptr,y3ptr,n,group_results_gpu);

        // copy results back to
        cuda_ierr = hipMemcpy(group_results_cpu,group_results_gpu,sizeof(PetscScalar) * MDOT_WORKGROUP_NUM * 4,hipMemcpyDeviceToHost);CHKERRCUDA(cuda_ierr);

        // sum group results into z:
        for (j=0; j<4; ++j) {
          z[current_y_index + j] = 0;
          for (i=j*MDOT_WORKGROUP_NUM; i<(j+1)*MDOT_WORKGROUP_NUM; ++i) z[current_y_index + j] += group_results_cpu[i];
        }
#endif
        ierr = VecCUDARestoreArrayRead(yin[current_y_index  ],&y0ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+1],&y1ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+2],&y2ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+3],&y3ptr);CHKERRQ(ierr);
        current_y_index += 4;
        break;

      case 3:
        ierr = VecCUDAGetArrayRead(yin[current_y_index  ],&y0ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+1],&y1ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+2],&y2ptr);CHKERRQ(ierr);

#if defined(PETSC_USE_COMPLEX)
        cberr = cublasXdot(cublasv2handle,bn,y0ptr,one,xptr,one,&z[current_y_index]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y1ptr,one,xptr,one,&z[current_y_index+1]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y2ptr,one,xptr,one,&z[current_y_index+2]);CHKERRCUBLAS(cberr);
#else
        // run kernel:
        VecMDot_SeqCUDA_kernel3<<<MDOT_WORKGROUP_NUM,MDOT_WORKGROUP_SIZE>>>(xptr,y0ptr,y1ptr,y2ptr,n,group_results_gpu);

        // copy results back to
        cuda_ierr = hipMemcpy(group_results_cpu,group_results_gpu,sizeof(PetscScalar) * MDOT_WORKGROUP_NUM * 3,hipMemcpyDeviceToHost);CHKERRCUDA(cuda_ierr);

        // sum group results into z:
        for (j=0; j<3; ++j) {
          z[current_y_index + j] = 0;
          for (i=j*MDOT_WORKGROUP_NUM; i<(j+1)*MDOT_WORKGROUP_NUM; ++i) z[current_y_index + j] += group_results_cpu[i];
        }
#endif

        ierr = VecCUDARestoreArrayRead(yin[current_y_index  ],&y0ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+1],&y1ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+2],&y2ptr);CHKERRQ(ierr);
        current_y_index += 3;
        break;

      case 2:
        ierr = VecCUDAGetArrayRead(yin[current_y_index],&y0ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+1],&y1ptr);CHKERRQ(ierr);

#if defined(PETSC_USE_COMPLEX)
        cberr = cublasXdot(cublasv2handle,bn,y0ptr,one,xptr,one,&z[current_y_index]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y1ptr,one,xptr,one,&z[current_y_index+1]);CHKERRCUBLAS(cberr);
#else
        // run kernel:
        VecMDot_SeqCUDA_kernel2<<<MDOT_WORKGROUP_NUM,MDOT_WORKGROUP_SIZE>>>(xptr,y0ptr,y1ptr,n,group_results_gpu);

        // copy results back to
        cuda_ierr = hipMemcpy(group_results_cpu,group_results_gpu,sizeof(PetscScalar) * MDOT_WORKGROUP_NUM * 2,hipMemcpyDeviceToHost);CHKERRCUDA(cuda_ierr);

        // sum group results into z:
        for (j=0; j<2; ++j) {
          z[current_y_index + j] = 0;
          for (i=j*MDOT_WORKGROUP_NUM; i<(j+1)*MDOT_WORKGROUP_NUM; ++i) z[current_y_index + j] += group_results_cpu[i];
        }
#endif
        ierr = VecCUDARestoreArrayRead(yin[current_y_index],&y0ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+1],&y1ptr);CHKERRQ(ierr);
        current_y_index += 2;
        break;

      case 1:
        ierr = VecCUDAGetArrayRead(yin[current_y_index],&y0ptr);CHKERRQ(ierr);
        cberr = cublasXdot(cublasv2handle,bn,y0ptr,one,xptr,one,&z[current_y_index]);CHKERRCUBLAS(cberr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index],&y0ptr);CHKERRQ(ierr);
        current_y_index += 1;
        break;

      default: // 8 or more vectors left
        ierr = VecCUDAGetArrayRead(yin[current_y_index  ],&y0ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+1],&y1ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+2],&y2ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+3],&y3ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+4],&y4ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+5],&y5ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+6],&y6ptr);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayRead(yin[current_y_index+7],&y7ptr);CHKERRQ(ierr);

#if defined(PETSC_USE_COMPLEX)
        cberr = cublasXdot(cublasv2handle,bn,y0ptr,one,xptr,one,&z[current_y_index]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y1ptr,one,xptr,one,&z[current_y_index+1]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y2ptr,one,xptr,one,&z[current_y_index+2]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y3ptr,one,xptr,one,&z[current_y_index+3]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y4ptr,one,xptr,one,&z[current_y_index+4]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y5ptr,one,xptr,one,&z[current_y_index+5]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y6ptr,one,xptr,one,&z[current_y_index+6]);CHKERRCUBLAS(cberr);
        cberr = cublasXdot(cublasv2handle,bn,y7ptr,one,xptr,one,&z[current_y_index+7]);CHKERRCUBLAS(cberr);
#else
        // run kernel:
        VecMDot_SeqCUDA_kernel8<<<MDOT_WORKGROUP_NUM,MDOT_WORKGROUP_SIZE>>>(xptr,y0ptr,y1ptr,y2ptr,y3ptr,y4ptr,y5ptr,y6ptr,y7ptr,n,group_results_gpu);

        // copy results back to
        cuda_ierr = hipMemcpy(group_results_cpu,group_results_gpu,sizeof(PetscScalar) * MDOT_WORKGROUP_NUM * 8,hipMemcpyDeviceToHost);CHKERRCUDA(cuda_ierr);

        // sum group results into z:
        for (j=0; j<8; ++j) {
          z[current_y_index + j] = 0;
          for (i=j*MDOT_WORKGROUP_NUM; i<(j+1)*MDOT_WORKGROUP_NUM; ++i) z[current_y_index + j] += group_results_cpu[i];
        }
#endif
        ierr = VecCUDARestoreArrayRead(yin[current_y_index  ],&y0ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+1],&y1ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+2],&y2ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+3],&y3ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+4],&y4ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+5],&y5ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+6],&y6ptr);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayRead(yin[current_y_index+7],&y7ptr);CHKERRQ(ierr);
        current_y_index += 8;
        break;
    }
  }
  ierr = VecCUDARestoreArrayRead(xin,&xptr);CHKERRQ(ierr);

  cuda_ierr = hipFree(group_results_gpu);CHKERRCUDA(cuda_ierr);
  ierr = PetscLogFlops(PetscMax(nv*(2.0*n-1),0.0));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef MDOT_WORKGROUP_SIZE
#undef MDOT_WORKGROUP_NUM

PetscErrorCode VecSet_SeqCUDA(Vec xin,PetscScalar alpha)
{
  PetscInt                        n = xin->map->n;
  PetscScalar                     *xarray=NULL;
  thrust::device_ptr<PetscScalar> xptr;
  PetscErrorCode                  ierr;
  hipError_t                     err;

  PetscFunctionBegin;
  ierr = VecCUDAGetArrayWrite(xin,&xarray);CHKERRQ(ierr);
  if (alpha == (PetscScalar)0.0) {
    err = hipMemset(xarray,0,n*sizeof(PetscScalar));CHKERRCUDA(err);
  } else {
    try {
      xptr = thrust::device_pointer_cast(xarray);
      thrust::fill(xptr,xptr+n,alpha);
    } catch (char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Thrust error: %s", ex);
    }
  }
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  ierr = VecCUDARestoreArrayWrite(xin,&xarray);
  PetscFunctionReturn(0);
}

PetscErrorCode VecScale_SeqCUDA(Vec xin,PetscScalar alpha)
{
  PetscScalar    *xarray;
  PetscErrorCode ierr;
  PetscBLASInt   one=1,bn;
  hipblasHandle_t cublasv2handle;
  hipblasStatus_t cberr;

  PetscFunctionBegin;
  if (alpha == (PetscScalar)0.0) {
    ierr = VecSet_SeqCUDA(xin,alpha);CHKERRQ(ierr);
  } else if (alpha != (PetscScalar)1.0) {
    ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
    ierr = PetscBLASIntCast(xin->map->n,&bn);CHKERRQ(ierr);
    ierr = VecCUDAGetArrayReadWrite(xin,&xarray);CHKERRQ(ierr);
    cberr = cublasXscal(cublasv2handle,bn,&alpha,xarray,one);CHKERRCUBLAS(cberr);
    ierr = VecCUDARestoreArrayReadWrite(xin,&xarray);CHKERRQ(ierr);
  }
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  ierr = PetscLogFlops(xin->map->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecTDot_SeqCUDA(Vec xin,Vec yin,PetscScalar *z)
{
  const PetscScalar *xarray,*yarray;
  PetscErrorCode    ierr;
  PetscBLASInt      one=1,bn;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    cberr;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(xin->map->n,&bn);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(yin,&yarray);CHKERRQ(ierr);
  cberr = cublasXdotu(cublasv2handle,bn,xarray,one,yarray,one,z);CHKERRCUBLAS(cberr);
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  if (xin->map->n > 0) {
    ierr = PetscLogFlops(2.0*xin->map->n-1);CHKERRQ(ierr);
  }
  ierr = VecCUDARestoreArrayRead(yin,&yarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecCopy_SeqCUDA(Vec xin,Vec yin)
{
  const PetscScalar *xarray;
  PetscScalar       *yarray;
  PetscErrorCode    ierr;
  hipError_t       err;

  PetscFunctionBegin;
  if (xin != yin) {
    if (xin->valid_GPU_array == PETSC_OFFLOAD_GPU) {
      ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
      ierr = VecCUDAGetArrayWrite(yin,&yarray);CHKERRQ(ierr);
      err = hipMemcpy(yarray,xarray,yin->map->n*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(err);
      ierr = WaitForGPU();CHKERRCUDA(ierr);
      ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
      ierr = VecCUDARestoreArrayWrite(yin,&yarray);CHKERRQ(ierr);

    } else if (xin->valid_GPU_array == PETSC_OFFLOAD_CPU) {
      /* copy in CPU if we are on the CPU*/
      ierr = VecCopy_SeqCUDA_Private(xin,yin);CHKERRQ(ierr);
    } else if (xin->valid_GPU_array == PETSC_OFFLOAD_BOTH) {
      /* if xin is valid in both places, see where yin is and copy there (because it's probably where we'll want to next use it) */
      if (yin->valid_GPU_array == PETSC_OFFLOAD_CPU) {
        /* copy in CPU */
        ierr = VecCopy_SeqCUDA_Private(xin,yin);CHKERRQ(ierr);

      } else if (yin->valid_GPU_array == PETSC_OFFLOAD_GPU) {
        /* copy in GPU */
        ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayWrite(yin,&yarray);CHKERRQ(ierr);
        err = hipMemcpy(yarray,xarray,yin->map->n*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(err);
        ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayWrite(yin,&yarray);CHKERRQ(ierr);
      } else if (yin->valid_GPU_array == PETSC_OFFLOAD_BOTH) {
        /* xin and yin are both valid in both places (or yin was unallocated before the earlier call to allocatecheck
           default to copy in GPU (this is an arbitrary choice) */
        ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
        ierr = VecCUDAGetArrayWrite(yin,&yarray);CHKERRQ(ierr);
        err = hipMemcpy(yarray,xarray,yin->map->n*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(err);
        ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
        ierr = VecCUDARestoreArrayWrite(yin,&yarray);CHKERRQ(ierr);
      } else {
        ierr = VecCopy_SeqCUDA_Private(xin,yin);CHKERRQ(ierr);
      }
    }
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecSwap_SeqCUDA(Vec xin,Vec yin)
{
  PetscErrorCode ierr;
  PetscBLASInt   one = 1,bn;
  PetscScalar    *xarray,*yarray;
  hipblasHandle_t cublasv2handle;
  hipblasStatus_t cberr;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(xin->map->n,&bn);CHKERRQ(ierr);
  if (xin != yin) {
    ierr = VecCUDAGetArrayReadWrite(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDAGetArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
    cberr = cublasXswap(cublasv2handle,bn,xarray,one,yarray,one);CHKERRCUBLAS(cberr);
    ierr = WaitForGPU();CHKERRCUDA(ierr);
    ierr = VecCUDARestoreArrayReadWrite(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDARestoreArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecAXPBY_SeqCUDA(Vec yin,PetscScalar alpha,PetscScalar beta,Vec xin)
{
  PetscErrorCode    ierr;
  PetscScalar       a = alpha,b = beta;
  const PetscScalar *xarray;
  PetscScalar       *yarray;
  PetscBLASInt      one = 1, bn;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    cberr;
  hipError_t       err;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(yin->map->n,&bn);CHKERRQ(ierr);
  if (a == (PetscScalar)0.0) {
    ierr = VecScale_SeqCUDA(yin,beta);CHKERRQ(ierr);
  } else if (b == (PetscScalar)1.0) {
    ierr = VecAXPY_SeqCUDA(yin,alpha,xin);CHKERRQ(ierr);
  } else if (a == (PetscScalar)1.0) {
    ierr = VecAYPX_SeqCUDA(yin,beta,xin);CHKERRQ(ierr);
  } else if (b == (PetscScalar)0.0) {
    ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDAGetArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
    err = hipMemcpy(yarray,xarray,yin->map->n*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(err);
    cberr = cublasXscal(cublasv2handle,bn,&alpha,yarray,one);CHKERRCUBLAS(cberr);
    ierr = PetscLogFlops(xin->map->n);CHKERRQ(ierr);
    ierr = WaitForGPU();CHKERRCUDA(ierr);
    ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDARestoreArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
  } else {
    ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDAGetArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
    cberr = cublasXscal(cublasv2handle,bn,&beta,yarray,one);CHKERRCUBLAS(cberr);
    cberr = cublasXaxpy(cublasv2handle,bn,&alpha,xarray,one,yarray,one);CHKERRCUBLAS(cberr);
    ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = VecCUDARestoreArrayReadWrite(yin,&yarray);CHKERRQ(ierr);
    ierr = WaitForGPU();CHKERRCUDA(ierr);
    ierr = PetscLogFlops(3.0*xin->map->n);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecAXPBYPCZ_SeqCUDA(Vec zin,PetscScalar alpha,PetscScalar beta,PetscScalar gamma,Vec xin,Vec yin)
{
  PetscErrorCode ierr;
  PetscInt       n = zin->map->n;

  PetscFunctionBegin;
  if (gamma == (PetscScalar)1.0) {
    /* z = ax + b*y + z */
    ierr = VecAXPY_SeqCUDA(zin,alpha,xin);CHKERRQ(ierr);
    ierr = VecAXPY_SeqCUDA(zin,beta,yin);CHKERRQ(ierr);
    ierr = PetscLogFlops(4.0*n);CHKERRQ(ierr);
  } else {
    /* z = a*x + b*y + c*z */
    ierr = VecScale_SeqCUDA(zin,gamma);CHKERRQ(ierr);
    ierr = VecAXPY_SeqCUDA(zin,alpha,xin);CHKERRQ(ierr);
    ierr = VecAXPY_SeqCUDA(zin,beta,yin);CHKERRQ(ierr);
    ierr = PetscLogFlops(5.0*n);CHKERRQ(ierr);
  }
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecPointwiseMult_SeqCUDA(Vec win,Vec xin,Vec yin)
{
  PetscInt                              n = win->map->n;
  const PetscScalar                     *xarray,*yarray;
  PetscScalar                           *warray;
  thrust::device_ptr<const PetscScalar> xptr,yptr;
  thrust::device_ptr<PetscScalar>       wptr;
  PetscErrorCode                        ierr;

  PetscFunctionBegin;
  ierr = VecCUDAGetArrayReadWrite(win,&warray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(yin,&yarray);CHKERRQ(ierr);
  try {
    wptr = thrust::device_pointer_cast(warray);
    xptr = thrust::device_pointer_cast(xarray);
    yptr = thrust::device_pointer_cast(yarray);
    thrust::transform(xptr,xptr+n,yptr,wptr,thrust::multiplies<PetscScalar>());
    ierr = WaitForGPU();CHKERRCUDA(ierr);
  } catch (char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Thrust error: %s", ex);
  }
  ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(yin,&yarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayReadWrite(win,&warray);CHKERRQ(ierr);
  ierr = PetscLogFlops(n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/* should do infinity norm in cuda */

PetscErrorCode VecNorm_SeqCUDA(Vec xin,NormType type,PetscReal *z)
{
  PetscErrorCode    ierr;
  PetscInt          n = xin->map->n;
  PetscBLASInt      one = 1, bn;
  const PetscScalar *xarray;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    cberr;
  hipError_t       err;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(n,&bn);CHKERRQ(ierr);
  if (type == NORM_2 || type == NORM_FROBENIUS) {
    ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    cberr = cublasXnrm2(cublasv2handle,bn,xarray,one,z);CHKERRCUBLAS(cberr);
    ierr = WaitForGPU();CHKERRCUDA(ierr);
    ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = PetscLogFlops(PetscMax(2.0*n-1,0.0));CHKERRQ(ierr);
  } else if (type == NORM_INFINITY) {
    int  i;
    ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    cberr = cublasIXamax(cublasv2handle,bn,xarray,one,&i);CHKERRCUBLAS(cberr);
    if (bn) {
      PetscScalar zs;

      err = hipMemcpy(&zs,xarray+i-1,sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(err);
      *z = PetscAbsScalar(zs);
    } else *z = 0.0;
    ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
  } else if (type == NORM_1) {
    ierr = VecCUDAGetArrayRead(xin,&xarray);CHKERRQ(ierr);
    cberr = cublasXasum(cublasv2handle,bn,xarray,one,z);CHKERRCUBLAS(cberr);
    ierr = VecCUDARestoreArrayRead(xin,&xarray);CHKERRQ(ierr);
    ierr = WaitForGPU();CHKERRCUDA(ierr);
    ierr = PetscLogFlops(PetscMax(n-1.0,0.0));CHKERRQ(ierr);
  } else if (type == NORM_1_AND_2) {
    ierr = VecNorm_SeqCUDA(xin,NORM_1,z);CHKERRQ(ierr);
    ierr = VecNorm_SeqCUDA(xin,NORM_2,z+1);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecDotNorm2_SeqCUDA(Vec s, Vec t, PetscScalar *dp, PetscScalar *nm)
{
  PetscErrorCode    ierr;
  PetscReal         n=s->map->n;
  const PetscScalar *sarray,*tarray;

  PetscFunctionBegin;
  ierr = VecCUDAGetArrayRead(s,&sarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(t,&tarray);CHKERRQ(ierr);
  ierr = VecDot_SeqCUDA(s,t,dp);CHKERRQ(ierr);
  ierr = VecDot_SeqCUDA(t,t,nm);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(s,&sarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(t,&tarray);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  ierr = PetscLogFlops(4.0*n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecDestroy_SeqCUDA(Vec v)
{
  PetscErrorCode ierr;
  hipError_t    err;

  PetscFunctionBegin;
  if (v->spptr) {
    if (((Vec_CUDA*)v->spptr)->GPUarray_allocated) {
      err = hipFree(((Vec_CUDA*)v->spptr)->GPUarray_allocated);CHKERRCUDA(err);
      ((Vec_CUDA*)v->spptr)->GPUarray_allocated = NULL;
    }
    if (((Vec_CUDA*)v->spptr)->stream) {
      err = hipStreamDestroy(((Vec_CUDA*)v->spptr)->stream);CHKERRCUDA(err);
    }
    ierr = PetscFree(v->spptr);CHKERRQ(ierr);
  }
  ierr = VecDestroy_SeqCUDA_Private(v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#if defined(PETSC_USE_COMPLEX)
struct conjugate
{
  __host__ __device__
    PetscScalar operator()(PetscScalar x)
    {
      return PetscConj(x);
    }
};
#endif

PetscErrorCode VecConjugate_SeqCUDA(Vec xin)
{
  PetscScalar                     *xarray;
  PetscErrorCode                  ierr;
#if defined(PETSC_USE_COMPLEX)
  PetscInt                        n = xin->map->n;
  thrust::device_ptr<PetscScalar> xptr;
#endif

  PetscFunctionBegin;
  ierr = VecCUDAGetArrayReadWrite(xin,&xarray);CHKERRQ(ierr);
#if defined(PETSC_USE_COMPLEX)
  try {
    xptr = thrust::device_pointer_cast(xarray);
    thrust::transform(xptr,xptr+n,xptr,conjugate());
    ierr = WaitForGPU();CHKERRCUDA(ierr);
  } catch (char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Thrust error: %s", ex);
  }
#endif
  ierr = VecCUDARestoreArrayReadWrite(xin,&xarray);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecGetLocalVector_SeqCUDA(Vec v,Vec w)
{
  PetscErrorCode ierr;
  hipError_t    err;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(v,VEC_CLASSID,1);
  PetscValidHeaderSpecific(w,VEC_CLASSID,2);
  PetscCheckTypeName(w,VECSEQCUDA);

  if (w->data) {
    if (((Vec_Seq*)w->data)->array_allocated) {
      ierr = PetscFree(((Vec_Seq*)w->data)->array_allocated);CHKERRQ(ierr);
    }
    ((Vec_Seq*)w->data)->array = NULL;
    ((Vec_Seq*)w->data)->unplacedarray = NULL;
  }
  if (w->spptr) {
    if (((Vec_CUDA*)w->spptr)->GPUarray) {
      err = hipFree(((Vec_CUDA*)w->spptr)->GPUarray);CHKERRCUDA(err);
      ((Vec_CUDA*)w->spptr)->GPUarray = NULL;
    }
    err = hipStreamDestroy(((Vec_CUDA*)w->spptr)->stream);CHKERRCUDA(err);
    ierr = PetscFree(w->spptr);CHKERRQ(ierr);
  }

  if (v->petscnative) {
    ierr = PetscFree(w->data);CHKERRQ(ierr);
    w->data = v->data;
    w->valid_GPU_array = v->valid_GPU_array;
    w->spptr = v->spptr;
    ierr = PetscObjectStateIncrease((PetscObject)w);CHKERRQ(ierr);
  } else {
    ierr = VecGetArray(v,&((Vec_Seq*)w->data)->array);CHKERRQ(ierr);
    w->valid_GPU_array = PETSC_OFFLOAD_CPU;
    ierr = VecCUDAAllocateCheck(w);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecRestoreLocalVector_SeqCUDA(Vec v,Vec w)
{
  PetscErrorCode ierr;
  hipError_t    err;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(v,VEC_CLASSID,1);
  PetscValidHeaderSpecific(w,VEC_CLASSID,2);
  PetscCheckTypeName(w,VECSEQCUDA);

  if (v->petscnative) {
    v->data = w->data;
    v->valid_GPU_array = w->valid_GPU_array;
    v->spptr = w->spptr;
    ierr = VecCUDACopyFromGPU(v);CHKERRQ(ierr);
    ierr = PetscObjectStateIncrease((PetscObject)v);CHKERRQ(ierr);
    w->data = 0;
    w->valid_GPU_array = PETSC_OFFLOAD_UNALLOCATED;
    w->spptr = 0;
  } else {
    ierr = VecRestoreArray(v,&((Vec_Seq*)w->data)->array);CHKERRQ(ierr);
    if ((Vec_CUDA*)w->spptr) {
      err = hipFree(((Vec_CUDA*)w->spptr)->GPUarray);CHKERRCUDA(err);
      ((Vec_CUDA*)w->spptr)->GPUarray = NULL;
      err = hipStreamDestroy(((Vec_CUDA*)w->spptr)->stream);CHKERRCUDA(err);
      ierr = PetscFree(w->spptr);CHKERRQ(ierr);
    }
  }
  PetscFunctionReturn(0);
}

/*@C
   VecCUDAGetArrayReadWrite - Provides access to the CUDA buffer inside a vector.

   This function has semantics similar to VecGetArray():  the pointer
   returned by this function points to a consistent view of the vector
   data.  This may involve a copy operation of data from the host to the
   device if the data on the device is out of date.  If the device
   memory hasn't been allocated previously it will be allocated as part
   of this function call.  VecCUDAGetArrayReadWrite() assumes that
   the user will modify the vector data.  This is similar to
   intent(inout) in fortran.

   The CUDA device pointer has to be released by calling
   VecCUDARestoreArrayReadWrite().  Upon restoring the vector data
   the data on the host will be marked as out of date.  A subsequent
   access of the host data will thus incur a data transfer from the
   device to the host.


   Input Parameter:
.  v - the vector

   Output Parameter:
.  a - the CUDA device pointer

   Fortran note:
   This function is not currently available from Fortran.

   Level: intermediate

.seealso: VecCUDARestoreArrayReadWrite(), VecCUDAGetArrayRead(), VecCUDAGetArrayWrite(), VecGetArray(), VecGetArrayRead()
@*/
PETSC_EXTERN PetscErrorCode VecCUDAGetArrayReadWrite(Vec v, PetscScalar **a)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscCheckTypeNames(v,VECSEQCUDA,VECMPICUDA);
  *a   = 0;
  ierr = VecCUDACopyToGPU(v);CHKERRQ(ierr);
  *a   = ((Vec_CUDA*)v->spptr)->GPUarray;
  PetscFunctionReturn(0);
}

/*@C
   VecCUDARestoreArrayReadWrite - Restore a CUDA device pointer previously acquired with VecCUDAGetArrayReadWrite().

   This marks the host data as out of date.  Subsequent access to the
   vector data on the host side with for instance VecGetArray() incurs a
   data transfer.

   Input Parameter:
+  v - the vector
-  a - the CUDA device pointer.  This pointer is invalid after
       VecCUDARestoreArrayReadWrite() returns.

   Fortran note:
   This function is not currently available from Fortran.

   Level: intermediate

.seealso: VecCUDAGetArrayReadWrite(), VecCUDAGetArrayRead(), VecCUDAGetArrayWrite(), VecGetArray(), VecRestoreArray(), VecGetArrayRead()
@*/
PETSC_EXTERN PetscErrorCode VecCUDARestoreArrayReadWrite(Vec v, PetscScalar **a)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscCheckTypeNames(v,VECSEQCUDA,VECMPICUDA);
  v->valid_GPU_array = PETSC_OFFLOAD_GPU;

  ierr = PetscObjectStateIncrease((PetscObject)v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*@C
   VecCUDAGetArrayRead - Provides read access to the CUDA buffer inside a vector.

   This function is analogous to VecGetArrayRead():  The pointer
   returned by this function points to a consistent view of the vector
   data.  This may involve a copy operation of data from the host to the
   device if the data on the device is out of date.  If the device
   memory hasn't been allocated previously it will be allocated as part
   of this function call.  VecCUDAGetArrayRead() assumes that the
   user will not modify the vector data.  This is analgogous to
   intent(in) in Fortran.

   The CUDA device pointer has to be released by calling
   VecCUDARestoreArrayRead().  If the data on the host side was
   previously up to date it will remain so, i.e. data on both the device
   and the host is up to date.  Accessing data on the host side does not
   incur a device to host data transfer.

   Input Parameter:
.  v - the vector

   Output Parameter:
.  a - the CUDA pointer.

   Fortran note:
   This function is not currently available from Fortran.

   Level: intermediate

.seealso: VecCUDARestoreArrayRead(), VecCUDAGetArrayReadWrite(), VecCUDAGetArrayWrite(), VecGetArray(), VecGetArrayRead()
@*/
PETSC_EXTERN PetscErrorCode VecCUDAGetArrayRead(Vec v, const PetscScalar **a)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscCheckTypeNames(v,VECSEQCUDA,VECMPICUDA);
  *a   = 0;
  ierr = VecCUDACopyToGPU(v);CHKERRQ(ierr);
  *a   = ((Vec_CUDA*)v->spptr)->GPUarray;
  PetscFunctionReturn(0);
}

/*@C
   VecCUDARestoreArrayRead - Restore a CUDA device pointer previously acquired with VecCUDAGetArrayRead().

   If the data on the host side was previously up to date it will remain
   so, i.e. data on both the device and the host is up to date.
   Accessing data on the host side e.g. with VecGetArray() does not
   incur a device to host data transfer.

   Input Parameter:
+  v - the vector
-  a - the CUDA device pointer.  This pointer is invalid after
       VecCUDARestoreArrayRead() returns.

   Fortran note:
   This function is not currently available from Fortran.

   Level: intermediate

.seealso: VecCUDAGetArrayRead(), VecCUDAGetArrayWrite(), VecCUDAGetArrayReadWrite(), VecGetArray(), VecRestoreArray(), VecGetArrayRead()
@*/
PETSC_EXTERN PetscErrorCode VecCUDARestoreArrayRead(Vec v, const PetscScalar **a)
{
  PetscFunctionBegin;
  PetscCheckTypeNames(v,VECSEQCUDA,VECMPICUDA);
  PetscFunctionReturn(0);
}

/*@C
   VecCUDAGetArrayWrite - Provides write access to the CUDA buffer inside a vector.

   The data pointed to by the device pointer is uninitialized.  The user
   may not read from this data.  Furthermore, the entire array needs to
   be filled by the user to obtain well-defined behaviour.  The device
   memory will be allocated by this function if it hasn't been allocated
   previously.  This is analogous to intent(out) in Fortran.

   The device pointer needs to be released with
   VecCUDARestoreArrayWrite().  When the pointer is released the
   host data of the vector is marked as out of data.  Subsequent access
   of the host data with e.g. VecGetArray() incurs a device to host data
   transfer.


   Input Parameter:
.  v - the vector

   Output Parameter:
.  a - the CUDA pointer

   Fortran note:
   This function is not currently available from Fortran.

   Level: advanced

.seealso: VecCUDARestoreArrayWrite(), VecCUDAGetArrayReadWrite(), VecCUDAGetArrayRead(), VecCUDAGetArrayWrite(), VecGetArray(), VecGetArrayRead()
@*/
PETSC_EXTERN PetscErrorCode VecCUDAGetArrayWrite(Vec v, PetscScalar **a)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscCheckTypeNames(v,VECSEQCUDA,VECMPICUDA);
  *a   = 0;
  ierr = VecCUDAAllocateCheck(v);CHKERRQ(ierr);
  *a   = ((Vec_CUDA*)v->spptr)->GPUarray;
  PetscFunctionReturn(0);
}

/*@C
   VecCUDARestoreArrayWrite - Restore a CUDA device pointer previously acquired with VecCUDAGetArrayWrite().

   Data on the host will be marked as out of date.  Subsequent access of
   the data on the host side e.g. with VecGetArray() will incur a device
   to host data transfer.

   Input Parameter:
+  v - the vector
-  a - the CUDA device pointer.  This pointer is invalid after
       VecCUDARestoreArrayWrite() returns.

   Fortran note:
   This function is not currently available from Fortran.

   Level: intermediate

.seealso: VecCUDAGetArrayWrite(), VecCUDAGetArrayReadWrite(), VecCUDAGetArrayRead(), VecCUDAGetArrayWrite(), VecGetArray(), VecRestoreArray(), VecGetArrayRead()
@*/
PETSC_EXTERN PetscErrorCode VecCUDARestoreArrayWrite(Vec v, PetscScalar **a)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscCheckTypeNames(v,VECSEQCUDA,VECMPICUDA);
  v->valid_GPU_array = PETSC_OFFLOAD_GPU;

  ierr = PetscObjectStateIncrease((PetscObject)v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*@C
   VecCUDAPlaceArray - Allows one to replace the GPU array in a vector with a
   GPU array provided by the user. This is useful to avoid copying an
   array into a vector.

   Not Collective

   Input Parameters:
+  vec - the vector
-  array - the GPU array

   Notes:
   You can return to the original GPU array with a call to VecCUDAResetArray()
   It is not possible to use VecCUDAPlaceArray() and VecPlaceArray() at the
   same time on the same vector.

   Level: developer

.seealso: VecPlaceArray(), VecGetArray(), VecRestoreArray(), VecReplaceArray(), VecResetArray(), VecCUDAResetArray(), VecCUDAReplaceArray()

@*/
PetscErrorCode VecCUDAPlaceArray(Vec vin,PetscScalar *a)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscCheckTypeNames(vin,VECSEQCUDA,VECMPICUDA);
  ierr = VecCUDACopyToGPU(vin);CHKERRQ(ierr);
  if (((Vec_Seq*)vin->data)->unplacedarray) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"VecCUDAPlaceArray()/VecPlaceArray() was already called on this vector, without a call to VecCUDAResetArray()/VecResetArray()");
  ((Vec_Seq*)vin->data)->unplacedarray  = (PetscScalar *) ((Vec_CUDA*)vin->spptr)->GPUarray; /* save previous GPU array so reset can bring it back */
  ((Vec_CUDA*)vin->spptr)->GPUarray = a;
  vin->valid_GPU_array = PETSC_OFFLOAD_GPU;
  ierr = PetscObjectStateIncrease((PetscObject)vin);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*@C
   VecCUDAReplaceArray - Allows one to replace the GPU array in a vector
   with a GPU array provided by the user. This is useful to avoid copying
   a GPU array into a vector.

   Not Collective

   Input Parameters:
+  vec - the vector
-  array - the GPU array

   Notes:
   This permanently replaces the GPU array and frees the memory associated
   with the old GPU array.

   The memory passed in CANNOT be freed by the user. It will be freed
   when the vector is destroyed.

   Not supported from Fortran

   Level: developer

.seealso: VecGetArray(), VecRestoreArray(), VecPlaceArray(), VecResetArray(), VecCUDAResetArray(), VecCUDAPlaceArray(), VecReplaceArray()

@*/
PetscErrorCode VecCUDAReplaceArray(Vec vin,PetscScalar *a)
{
  hipError_t err;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscCheckTypeNames(vin,VECSEQCUDA,VECMPICUDA);
  err = hipFree(((Vec_CUDA*)vin->spptr)->GPUarray);CHKERRCUDA(err);
  ((Vec_CUDA*)vin->spptr)->GPUarray = a;
  vin->valid_GPU_array = PETSC_OFFLOAD_GPU;
  ierr = PetscObjectStateIncrease((PetscObject)vin);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*@C
   VecCUDAResetArray - Resets a vector to use its default memory. Call this
   after the use of VecCUDAPlaceArray().

   Not Collective

   Input Parameters:
.  vec - the vector

   Level: developer

.seealso: VecGetArray(), VecRestoreArray(), VecReplaceArray(), VecPlaceArray(), VecResetArray(), VecCUDAPlaceArray(), VecCUDAReplaceArray()

@*/
PetscErrorCode VecCUDAResetArray(Vec vin)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscCheckTypeNames(vin,VECSEQCUDA,VECMPICUDA);
  ierr = VecCUDACopyToGPU(vin);CHKERRQ(ierr);
  ((Vec_CUDA*)vin->spptr)->GPUarray = (PetscScalar *) ((Vec_Seq*)vin->data)->unplacedarray;
  ((Vec_Seq*)vin->data)->unplacedarray = 0;
  vin->valid_GPU_array = PETSC_OFFLOAD_GPU;
  ierr = PetscObjectStateIncrease((PetscObject)vin);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
