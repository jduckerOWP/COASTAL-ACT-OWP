/*
  Defines the basic matrix operations for the AIJ (compressed row)
  matrix storage format using the CUSPARSE library,
*/
#define PETSC_SKIP_SPINLOCK

#include <petscconf.h>
#include <../src/mat/impls/aij/seq/aij.h>          /*I "petscmat.h" I*/
#include <../src/mat/impls/sbaij/seq/sbaij.h>
#include <../src/vec/vec/impls/dvecimpl.h>
#include <petsc/private/vecimpl.h>
#undef VecType
#include <../src/mat/impls/aij/seq/seqcusparse/cusparsematimpl.h>

const char *const MatCUSPARSEStorageFormats[] = {"CSR","ELL","HYB","MatCUSPARSEStorageFormat","MAT_CUSPARSE_",0};

static PetscErrorCode MatICCFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,const MatFactorInfo*);
static PetscErrorCode MatCholeskyFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,const MatFactorInfo*);
static PetscErrorCode MatCholeskyFactorNumeric_SeqAIJCUSPARSE(Mat,Mat,const MatFactorInfo*);

static PetscErrorCode MatILUFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,IS,const MatFactorInfo*);
static PetscErrorCode MatLUFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,IS,const MatFactorInfo*);
static PetscErrorCode MatLUFactorNumeric_SeqAIJCUSPARSE(Mat,Mat,const MatFactorInfo*);

static PetscErrorCode MatSolve_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatSolve_SeqAIJCUSPARSE_NaturalOrdering(Mat,Vec,Vec);
static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering(Mat,Vec,Vec);
static PetscErrorCode MatSetFromOptions_SeqAIJCUSPARSE(PetscOptionItems *PetscOptionsObject,Mat);
static PetscErrorCode MatMult_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatMultAdd_SeqAIJCUSPARSE(Mat,Vec,Vec,Vec);
static PetscErrorCode MatMultTranspose_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatMultTransposeAdd_SeqAIJCUSPARSE(Mat,Vec,Vec,Vec);

static PetscErrorCode CsrMatrix_Destroy(CsrMatrix**);
static PetscErrorCode MatSeqAIJCUSPARSEMultStruct_Destroy(Mat_SeqAIJCUSPARSETriFactorStruct**);
static PetscErrorCode MatSeqAIJCUSPARSEMultStruct_Destroy(Mat_SeqAIJCUSPARSEMultStruct**,MatCUSPARSEStorageFormat);
static PetscErrorCode MatSeqAIJCUSPARSETriFactors_Destroy(Mat_SeqAIJCUSPARSETriFactors**);
static PetscErrorCode MatSeqAIJCUSPARSE_Destroy(Mat_SeqAIJCUSPARSE**);

PetscErrorCode MatCUSPARSESetStream(Mat A,const hipStream_t stream)
{
  hipsparseStatus_t   stat;
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  cusparsestruct->stream = stream;
  stat = hipsparseSetStream(cusparsestruct->handle,cusparsestruct->stream);CHKERRCUDA(stat);
  PetscFunctionReturn(0);
}

PetscErrorCode MatCUSPARSESetHandle(Mat A,const hipsparseHandle_t handle)
{
  hipsparseStatus_t   stat;
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  if (cusparsestruct->handle != handle) {
    if (cusparsestruct->handle) {
      stat = hipsparseDestroy(cusparsestruct->handle);CHKERRCUDA(stat);
    }
    cusparsestruct->handle = handle;
  }
  stat = hipsparseSetPointerMode(cusparsestruct->handle, HIPSPARSE_POINTER_MODE_DEVICE);CHKERRCUDA(stat);
  PetscFunctionReturn(0);
}

PetscErrorCode MatCUSPARSEClearHandle(Mat A)
{
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  PetscFunctionBegin;
  if (cusparsestruct->handle)
    cusparsestruct->handle = 0;
  PetscFunctionReturn(0);
}

PetscErrorCode MatFactorGetSolverType_seqaij_cusparse(Mat A,MatSolverType *type)
{
  PetscFunctionBegin;
  *type = MATSOLVERCUSPARSE;
  PetscFunctionReturn(0);
}

/*MC
  MATSOLVERCUSPARSE = "cusparse" - A matrix type providing triangular solvers for seq matrices
  on a single GPU of type, seqaijcusparse, aijcusparse, or seqaijcusp, aijcusp. Currently supported
  algorithms are ILU(k) and ICC(k). Typically, deeper factorizations (larger k) results in poorer
  performance in the triangular solves. Full LU, and Cholesky decompositions can be solved through the
  CUSPARSE triangular solve algorithm. However, the performance can be quite poor and thus these
  algorithms are not recommended. This class does NOT support direct solver operations.

  Level: beginner

.seealso: PCFactorSetMatSolverType(), MatSolverType, MatCreateSeqAIJCUSPARSE(), MATAIJCUSPARSE, MatCreateAIJCUSPARSE(), MatCUSPARSESetFormat(), MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
M*/

PETSC_EXTERN PetscErrorCode MatGetFactor_seqaijcusparse_cusparse(Mat A,MatFactorType ftype,Mat *B)
{
  PetscErrorCode ierr;
  PetscInt       n = A->rmap->n;

  PetscFunctionBegin;
  ierr = MatCreate(PetscObjectComm((PetscObject)A),B);CHKERRQ(ierr);
  (*B)->factortype = ftype;
  ierr = MatSetSizes(*B,n,n,n,n);CHKERRQ(ierr);
  ierr = MatSetType(*B,MATSEQAIJCUSPARSE);CHKERRQ(ierr);

  if (ftype == MAT_FACTOR_LU || ftype == MAT_FACTOR_ILU || ftype == MAT_FACTOR_ILUDT) {
    ierr = MatSetBlockSizesFromMats(*B,A,A);CHKERRQ(ierr);
    (*B)->ops->ilufactorsymbolic = MatILUFactorSymbolic_SeqAIJCUSPARSE;
    (*B)->ops->lufactorsymbolic  = MatLUFactorSymbolic_SeqAIJCUSPARSE;
  } else if (ftype == MAT_FACTOR_CHOLESKY || ftype == MAT_FACTOR_ICC) {
    (*B)->ops->iccfactorsymbolic      = MatICCFactorSymbolic_SeqAIJCUSPARSE;
    (*B)->ops->choleskyfactorsymbolic = MatCholeskyFactorSymbolic_SeqAIJCUSPARSE;
  } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Factor type not supported for CUSPARSE Matrix Types");

  ierr = MatSeqAIJSetPreallocation(*B,MAT_SKIP_ALLOCATION,NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)(*B),"MatFactorGetSolverType_C",MatFactorGetSolverType_seqaij_cusparse);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatCUSPARSESetFormat_SeqAIJCUSPARSE(Mat A,MatCUSPARSEFormatOperation op,MatCUSPARSEStorageFormat format)
{
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
#if CUDA_VERSION>=4020
  switch (op) {
  case MAT_CUSPARSE_MULT:
    cusparsestruct->format = format;
    break;
  case MAT_CUSPARSE_ALL:
    cusparsestruct->format = format;
    break;
  default:
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_SUP,"unsupported operation %d for MatCUSPARSEFormatOperation. MAT_CUSPARSE_MULT and MAT_CUSPARSE_ALL are currently supported.",op);
  }
#else
  if (format==MAT_CUSPARSE_ELL || format==MAT_CUSPARSE_HYB) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"ELL (Ellpack) and HYB (Hybrid) storage format require CUDA 4.2 or later.");
#endif
  PetscFunctionReturn(0);
}

/*@
   MatCUSPARSESetFormat - Sets the storage format of CUSPARSE matrices for a particular
   operation. Only the MatMult operation can use different GPU storage formats
   for MPIAIJCUSPARSE matrices.
   Not Collective

   Input Parameters:
+  A - Matrix of type SEQAIJCUSPARSE
.  op - MatCUSPARSEFormatOperation. SEQAIJCUSPARSE matrices support MAT_CUSPARSE_MULT and MAT_CUSPARSE_ALL. MPIAIJCUSPARSE matrices support MAT_CUSPARSE_MULT_DIAG, MAT_CUSPARSE_MULT_OFFDIAG, and MAT_CUSPARSE_ALL.
-  format - MatCUSPARSEStorageFormat (one of MAT_CUSPARSE_CSR, MAT_CUSPARSE_ELL, MAT_CUSPARSE_HYB. The latter two require CUDA 4.2)

   Output Parameter:

   Level: intermediate

.seealso: MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
@*/
PetscErrorCode MatCUSPARSESetFormat(Mat A,MatCUSPARSEFormatOperation op,MatCUSPARSEStorageFormat format)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(A, MAT_CLASSID,1);
  ierr = PetscTryMethod(A, "MatCUSPARSESetFormat_C",(Mat,MatCUSPARSEFormatOperation,MatCUSPARSEStorageFormat),(A,op,format));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSetFromOptions_SeqAIJCUSPARSE(PetscOptionItems *PetscOptionsObject,Mat A)
{
  PetscErrorCode           ierr;
  MatCUSPARSEStorageFormat format;
  PetscBool                flg;
  Mat_SeqAIJCUSPARSE       *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  ierr = PetscOptionsHead(PetscOptionsObject,"SeqAIJCUSPARSE options");CHKERRQ(ierr);
  if (A->factortype==MAT_FACTOR_NONE) {
    ierr = PetscOptionsEnum("-mat_cusparse_mult_storage_format","sets storage format of (seq)aijcusparse gpu matrices for SpMV",
                            "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)cusparsestruct->format,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
    if (flg) {
      ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_MULT,format);CHKERRQ(ierr);
    }
  }
  ierr = PetscOptionsEnum("-mat_cusparse_storage_format","sets storage format of (seq)aijcusparse gpu matrices for SpMV and TriSolve",
                          "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)cusparsestruct->format,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
  if (flg) {
    ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_ALL,format);CHKERRQ(ierr);
  }
  ierr = PetscOptionsTail();CHKERRQ(ierr);
  PetscFunctionReturn(0);

}

static PetscErrorCode MatILUFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS isrow,IS iscol,const MatFactorInfo *info)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatILUFactorSymbolic_SeqAIJ(B,A,isrow,iscol,info);CHKERRQ(ierr);
  B->ops->lufactornumeric = MatLUFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatLUFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS isrow,IS iscol,const MatFactorInfo *info)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatLUFactorSymbolic_SeqAIJ(B,A,isrow,iscol,info);CHKERRQ(ierr);
  B->ops->lufactornumeric = MatLUFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatICCFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS perm,const MatFactorInfo *info)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatICCFactorSymbolic_SeqAIJ(B,A,perm,info);CHKERRQ(ierr);
  B->ops->choleskyfactornumeric = MatCholeskyFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatCholeskyFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS perm,const MatFactorInfo *info)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCholeskyFactorSymbolic_SeqAIJ(B,A,perm,info);CHKERRQ(ierr);
  B->ops->choleskyfactornumeric = MatCholeskyFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEBuildILULowerTriMatrix(Mat A)
{
  Mat_SeqAIJ                        *a = (Mat_SeqAIJ*)A->data;
  PetscInt                          n = A->rmap->n;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  hipsparseStatus_t                  stat;
  const PetscInt                    *ai = a->i,*aj = a->j,*vi;
  const MatScalar                   *aa = a->a,*v;
  PetscInt                          *AiLo, *AjLo;
  PetscScalar                       *AALo;
  PetscInt                          i,nz, nzLower, offset, rowOffset;
  PetscErrorCode                    ierr;

  PetscFunctionBegin;
  if (A->valid_GPU_matrix == PETSC_OFFLOAD_UNALLOCATED || A->valid_GPU_matrix == PETSC_OFFLOAD_CPU) {
    try {
      /* first figure out the number of nonzeros in the lower triangular matrix including 1's on the diagonal. */
      nzLower=n+ai[n]-ai[1];

      /* Allocate Space for the lower triangular matrix */
      ierr = hipHostMalloc((void**) &AiLo, (n+1)*sizeof(PetscInt));CHKERRCUDA(ierr);
      ierr = hipHostMalloc((void**) &AjLo, nzLower*sizeof(PetscInt));CHKERRCUDA(ierr);
      ierr = hipHostMalloc((void**) &AALo, nzLower*sizeof(PetscScalar));CHKERRCUDA(ierr);

      /* Fill the lower triangular matrix */
      AiLo[0]  = (PetscInt) 0;
      AiLo[n]  = nzLower;
      AjLo[0]  = (PetscInt) 0;
      AALo[0]  = (MatScalar) 1.0;
      v        = aa;
      vi       = aj;
      offset   = 1;
      rowOffset= 1;
      for (i=1; i<n; i++) {
        nz = ai[i+1] - ai[i];
        /* additional 1 for the term on the diagonal */
        AiLo[i]    = rowOffset;
        rowOffset += nz+1;

        ierr = PetscMemcpy(&(AjLo[offset]), vi, nz*sizeof(PetscInt));CHKERRQ(ierr);
        ierr = PetscMemcpy(&(AALo[offset]), v, nz*sizeof(PetscScalar));CHKERRQ(ierr);

        offset      += nz;
        AjLo[offset] = (PetscInt) i;
        AALo[offset] = (MatScalar) 1.0;
        offset      += 1;

        v  += nz;
        vi += nz;
      }

      /* allocate space for the triangular factor information */
      loTriFactor = new Mat_SeqAIJCUSPARSETriFactorStruct;

      /* Create the matrix description */
      stat = hipsparseCreateMatDescr(&loTriFactor->descr);CHKERRCUDA(stat);
      stat = hipsparseSetMatIndexBase(loTriFactor->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUDA(stat);
      stat = hipsparseSetMatType(loTriFactor->descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR);CHKERRCUDA(stat);
      stat = hipsparseSetMatFillMode(loTriFactor->descr, HIPSPARSE_FILL_MODE_LOWER);CHKERRCUDA(stat);
      stat = hipsparseSetMatDiagType(loTriFactor->descr, HIPSPARSE_DIAG_TYPE_UNIT);CHKERRCUDA(stat);

      /* Create the solve analysis information */
      stat = cusparseCreateSolveAnalysisInfo(&loTriFactor->solveInfo);CHKERRCUDA(stat);

      /* set the operation */
      loTriFactor->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

      /* set the matrix */
      loTriFactor->csrMat = new CsrMatrix;
      loTriFactor->csrMat->num_rows = n;
      loTriFactor->csrMat->num_cols = n;
      loTriFactor->csrMat->num_entries = nzLower;

      loTriFactor->csrMat->row_offsets = new THRUSTINTARRAY32(n+1);
      loTriFactor->csrMat->row_offsets->assign(AiLo, AiLo+n+1);

      loTriFactor->csrMat->column_indices = new THRUSTINTARRAY32(nzLower);
      loTriFactor->csrMat->column_indices->assign(AjLo, AjLo+nzLower);

      loTriFactor->csrMat->values = new THRUSTARRAY(nzLower);
      loTriFactor->csrMat->values->assign(AALo, AALo+nzLower);

      /* perform the solve analysis */
      stat = cusparse_analysis(cusparseTriFactors->handle, loTriFactor->solveOp,
                               loTriFactor->csrMat->num_rows, loTriFactor->csrMat->num_entries, loTriFactor->descr,
                               loTriFactor->csrMat->values->data().get(), loTriFactor->csrMat->row_offsets->data().get(),
                               loTriFactor->csrMat->column_indices->data().get(), loTriFactor->solveInfo);CHKERRCUDA(stat);

      /* assign the pointer. Is this really necessary? */
      ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->loTriFactorPtr = loTriFactor;

      ierr = hipHostFree(AiLo);CHKERRCUDA(ierr);
      ierr = hipHostFree(AjLo);CHKERRCUDA(ierr);
      ierr = hipHostFree(AALo);CHKERRCUDA(ierr);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEBuildILUUpperTriMatrix(Mat A)
{
  Mat_SeqAIJ                        *a = (Mat_SeqAIJ*)A->data;
  PetscInt                          n = A->rmap->n;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  hipsparseStatus_t                  stat;
  const PetscInt                    *aj = a->j,*adiag = a->diag,*vi;
  const MatScalar                   *aa = a->a,*v;
  PetscInt                          *AiUp, *AjUp;
  PetscScalar                       *AAUp;
  PetscInt                          i,nz, nzUpper, offset;
  PetscErrorCode                    ierr;

  PetscFunctionBegin;
  if (A->valid_GPU_matrix == PETSC_OFFLOAD_UNALLOCATED || A->valid_GPU_matrix == PETSC_OFFLOAD_CPU) {
    try {
      /* next, figure out the number of nonzeros in the upper triangular matrix. */
      nzUpper = adiag[0]-adiag[n];

      /* Allocate Space for the upper triangular matrix */
      ierr = hipHostMalloc((void**) &AiUp, (n+1)*sizeof(PetscInt));CHKERRCUDA(ierr);
      ierr = hipHostMalloc((void**) &AjUp, nzUpper*sizeof(PetscInt));CHKERRCUDA(ierr);
      ierr = hipHostMalloc((void**) &AAUp, nzUpper*sizeof(PetscScalar));CHKERRCUDA(ierr);

      /* Fill the upper triangular matrix */
      AiUp[0]=(PetscInt) 0;
      AiUp[n]=nzUpper;
      offset = nzUpper;
      for (i=n-1; i>=0; i--) {
        v  = aa + adiag[i+1] + 1;
        vi = aj + adiag[i+1] + 1;

        /* number of elements NOT on the diagonal */
        nz = adiag[i] - adiag[i+1]-1;

        /* decrement the offset */
        offset -= (nz+1);

        /* first, set the diagonal elements */
        AjUp[offset] = (PetscInt) i;
        AAUp[offset] = (MatScalar)1./v[nz];
        AiUp[i]      = AiUp[i+1] - (nz+1);

        ierr = PetscMemcpy(&(AjUp[offset+1]), vi, nz*sizeof(PetscInt));CHKERRQ(ierr);
        ierr = PetscMemcpy(&(AAUp[offset+1]), v, nz*sizeof(PetscScalar));CHKERRQ(ierr);
      }

      /* allocate space for the triangular factor information */
      upTriFactor = new Mat_SeqAIJCUSPARSETriFactorStruct;

      /* Create the matrix description */
      stat = hipsparseCreateMatDescr(&upTriFactor->descr);CHKERRCUDA(stat);
      stat = hipsparseSetMatIndexBase(upTriFactor->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUDA(stat);
      stat = hipsparseSetMatType(upTriFactor->descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR);CHKERRCUDA(stat);
      stat = hipsparseSetMatFillMode(upTriFactor->descr, HIPSPARSE_FILL_MODE_UPPER);CHKERRCUDA(stat);
      stat = hipsparseSetMatDiagType(upTriFactor->descr, HIPSPARSE_DIAG_TYPE_NON_UNIT);CHKERRCUDA(stat);

      /* Create the solve analysis information */
      stat = cusparseCreateSolveAnalysisInfo(&upTriFactor->solveInfo);CHKERRCUDA(stat);

      /* set the operation */
      upTriFactor->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

      /* set the matrix */
      upTriFactor->csrMat = new CsrMatrix;
      upTriFactor->csrMat->num_rows = n;
      upTriFactor->csrMat->num_cols = n;
      upTriFactor->csrMat->num_entries = nzUpper;

      upTriFactor->csrMat->row_offsets = new THRUSTINTARRAY32(n+1);
      upTriFactor->csrMat->row_offsets->assign(AiUp, AiUp+n+1);

      upTriFactor->csrMat->column_indices = new THRUSTINTARRAY32(nzUpper);
      upTriFactor->csrMat->column_indices->assign(AjUp, AjUp+nzUpper);

      upTriFactor->csrMat->values = new THRUSTARRAY(nzUpper);
      upTriFactor->csrMat->values->assign(AAUp, AAUp+nzUpper);

      /* perform the solve analysis */
      stat = cusparse_analysis(cusparseTriFactors->handle, upTriFactor->solveOp,
                               upTriFactor->csrMat->num_rows, upTriFactor->csrMat->num_entries, upTriFactor->descr,
                               upTriFactor->csrMat->values->data().get(), upTriFactor->csrMat->row_offsets->data().get(),
                               upTriFactor->csrMat->column_indices->data().get(), upTriFactor->solveInfo);CHKERRCUDA(stat);

      /* assign the pointer. Is this really necessary? */
      ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->upTriFactorPtr = upTriFactor;

      ierr = hipHostFree(AiUp);CHKERRCUDA(ierr);
      ierr = hipHostFree(AjUp);CHKERRCUDA(ierr);
      ierr = hipHostFree(AAUp);CHKERRCUDA(ierr);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEILUAnalysisAndCopyToGPU(Mat A)
{
  PetscErrorCode               ierr;
  Mat_SeqAIJ                   *a                  = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  IS                           isrow = a->row,iscol = a->icol;
  PetscBool                    row_identity,col_identity;
  const PetscInt               *r,*c;
  PetscInt                     n = A->rmap->n;

  PetscFunctionBegin;
  ierr = MatSeqAIJCUSPARSEBuildILULowerTriMatrix(A);CHKERRQ(ierr);
  ierr = MatSeqAIJCUSPARSEBuildILUUpperTriMatrix(A);CHKERRQ(ierr);

  cusparseTriFactors->workVector = new THRUSTARRAY(n);
  cusparseTriFactors->nnz=a->nz;

  A->valid_GPU_matrix = PETSC_OFFLOAD_BOTH;
  /*lower triangular indices */
  ierr = ISGetIndices(isrow,&r);CHKERRQ(ierr);
  ierr = ISIdentity(isrow,&row_identity);CHKERRQ(ierr);
  if (!row_identity) {
    cusparseTriFactors->rpermIndices = new THRUSTINTARRAY(n);
    cusparseTriFactors->rpermIndices->assign(r, r+n);
  }
  ierr = ISRestoreIndices(isrow,&r);CHKERRQ(ierr);

  /*upper triangular indices */
  ierr = ISGetIndices(iscol,&c);CHKERRQ(ierr);
  ierr = ISIdentity(iscol,&col_identity);CHKERRQ(ierr);
  if (!col_identity) {
    cusparseTriFactors->cpermIndices = new THRUSTINTARRAY(n);
    cusparseTriFactors->cpermIndices->assign(c, c+n);
  }
  ierr = ISRestoreIndices(iscol,&c);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEBuildICCTriMatrices(Mat A)
{
  Mat_SeqAIJ                        *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  hipsparseStatus_t                  stat;
  PetscErrorCode                    ierr;
  PetscInt                          *AiUp, *AjUp;
  PetscScalar                       *AAUp;
  PetscScalar                       *AALo;
  PetscInt                          nzUpper = a->nz,n = A->rmap->n,i,offset,nz,j;
  Mat_SeqSBAIJ                      *b = (Mat_SeqSBAIJ*)A->data;
  const PetscInt                    *ai = b->i,*aj = b->j,*vj;
  const MatScalar                   *aa = b->a,*v;

  PetscFunctionBegin;
  if (A->valid_GPU_matrix == PETSC_OFFLOAD_UNALLOCATED || A->valid_GPU_matrix == PETSC_OFFLOAD_CPU) {
    try {
      /* Allocate Space for the upper triangular matrix */
      ierr = hipHostMalloc((void**) &AiUp, (n+1)*sizeof(PetscInt));CHKERRCUDA(ierr);
      ierr = hipHostMalloc((void**) &AjUp, nzUpper*sizeof(PetscInt));CHKERRCUDA(ierr);
      ierr = hipHostMalloc((void**) &AAUp, nzUpper*sizeof(PetscScalar));CHKERRCUDA(ierr);
      ierr = hipHostMalloc((void**) &AALo, nzUpper*sizeof(PetscScalar));CHKERRCUDA(ierr);

      /* Fill the upper triangular matrix */
      AiUp[0]=(PetscInt) 0;
      AiUp[n]=nzUpper;
      offset = 0;
      for (i=0; i<n; i++) {
        /* set the pointers */
        v  = aa + ai[i];
        vj = aj + ai[i];
        nz = ai[i+1] - ai[i] - 1; /* exclude diag[i] */

        /* first, set the diagonal elements */
        AjUp[offset] = (PetscInt) i;
        AAUp[offset] = (MatScalar)1.0/v[nz];
        AiUp[i]      = offset;
        AALo[offset] = (MatScalar)1.0/v[nz];

        offset+=1;
        if (nz>0) {
          ierr = PetscMemcpy(&(AjUp[offset]), vj, nz*sizeof(PetscInt));CHKERRQ(ierr);
          ierr = PetscMemcpy(&(AAUp[offset]), v, nz*sizeof(PetscScalar));CHKERRQ(ierr);
          for (j=offset; j<offset+nz; j++) {
            AAUp[j] = -AAUp[j];
            AALo[j] = AAUp[j]/v[nz];
          }
          offset+=nz;
        }
      }

      /* allocate space for the triangular factor information */
      upTriFactor = new Mat_SeqAIJCUSPARSETriFactorStruct;

      /* Create the matrix description */
      stat = hipsparseCreateMatDescr(&upTriFactor->descr);CHKERRCUDA(stat);
      stat = hipsparseSetMatIndexBase(upTriFactor->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUDA(stat);
      stat = hipsparseSetMatType(upTriFactor->descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR);CHKERRCUDA(stat);
      stat = hipsparseSetMatFillMode(upTriFactor->descr, HIPSPARSE_FILL_MODE_UPPER);CHKERRCUDA(stat);
      stat = hipsparseSetMatDiagType(upTriFactor->descr, HIPSPARSE_DIAG_TYPE_UNIT);CHKERRCUDA(stat);

      /* Create the solve analysis information */
      stat = cusparseCreateSolveAnalysisInfo(&upTriFactor->solveInfo);CHKERRCUDA(stat);

      /* set the operation */
      upTriFactor->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

      /* set the matrix */
      upTriFactor->csrMat = new CsrMatrix;
      upTriFactor->csrMat->num_rows = A->rmap->n;
      upTriFactor->csrMat->num_cols = A->cmap->n;
      upTriFactor->csrMat->num_entries = a->nz;

      upTriFactor->csrMat->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
      upTriFactor->csrMat->row_offsets->assign(AiUp, AiUp+A->rmap->n+1);

      upTriFactor->csrMat->column_indices = new THRUSTINTARRAY32(a->nz);
      upTriFactor->csrMat->column_indices->assign(AjUp, AjUp+a->nz);

      upTriFactor->csrMat->values = new THRUSTARRAY(a->nz);
      upTriFactor->csrMat->values->assign(AAUp, AAUp+a->nz);

      /* perform the solve analysis */
      stat = cusparse_analysis(cusparseTriFactors->handle, upTriFactor->solveOp,
                               upTriFactor->csrMat->num_rows, upTriFactor->csrMat->num_entries, upTriFactor->descr,
                               upTriFactor->csrMat->values->data().get(), upTriFactor->csrMat->row_offsets->data().get(),
                               upTriFactor->csrMat->column_indices->data().get(), upTriFactor->solveInfo);CHKERRCUDA(stat);

      /* assign the pointer. Is this really necessary? */
      ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->upTriFactorPtr = upTriFactor;

      /* allocate space for the triangular factor information */
      loTriFactor = new Mat_SeqAIJCUSPARSETriFactorStruct;

      /* Create the matrix description */
      stat = hipsparseCreateMatDescr(&loTriFactor->descr);CHKERRCUDA(stat);
      stat = hipsparseSetMatIndexBase(loTriFactor->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUDA(stat);
      stat = hipsparseSetMatType(loTriFactor->descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR);CHKERRCUDA(stat);
      stat = hipsparseSetMatFillMode(loTriFactor->descr, HIPSPARSE_FILL_MODE_UPPER);CHKERRCUDA(stat);
      stat = hipsparseSetMatDiagType(loTriFactor->descr, HIPSPARSE_DIAG_TYPE_NON_UNIT);CHKERRCUDA(stat);

      /* Create the solve analysis information */
      stat = cusparseCreateSolveAnalysisInfo(&loTriFactor->solveInfo);CHKERRCUDA(stat);

      /* set the operation */
      loTriFactor->solveOp = HIPSPARSE_OPERATION_TRANSPOSE;

      /* set the matrix */
      loTriFactor->csrMat = new CsrMatrix;
      loTriFactor->csrMat->num_rows = A->rmap->n;
      loTriFactor->csrMat->num_cols = A->cmap->n;
      loTriFactor->csrMat->num_entries = a->nz;

      loTriFactor->csrMat->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
      loTriFactor->csrMat->row_offsets->assign(AiUp, AiUp+A->rmap->n+1);

      loTriFactor->csrMat->column_indices = new THRUSTINTARRAY32(a->nz);
      loTriFactor->csrMat->column_indices->assign(AjUp, AjUp+a->nz);

      loTriFactor->csrMat->values = new THRUSTARRAY(a->nz);
      loTriFactor->csrMat->values->assign(AALo, AALo+a->nz);

      /* perform the solve analysis */
      stat = cusparse_analysis(cusparseTriFactors->handle, loTriFactor->solveOp,
                               loTriFactor->csrMat->num_rows, loTriFactor->csrMat->num_entries, loTriFactor->descr,
                               loTriFactor->csrMat->values->data().get(), loTriFactor->csrMat->row_offsets->data().get(),
                               loTriFactor->csrMat->column_indices->data().get(), loTriFactor->solveInfo);CHKERRCUDA(stat);

      /* assign the pointer. Is this really necessary? */
      ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->loTriFactorPtr = loTriFactor;

      A->valid_GPU_matrix = PETSC_OFFLOAD_BOTH;
      ierr = hipHostFree(AiUp);CHKERRCUDA(ierr);
      ierr = hipHostFree(AjUp);CHKERRCUDA(ierr);
      ierr = hipHostFree(AAUp);CHKERRCUDA(ierr);
      ierr = hipHostFree(AALo);CHKERRCUDA(ierr);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEICCAnalysisAndCopyToGPU(Mat A)
{
  PetscErrorCode               ierr;
  Mat_SeqAIJ                   *a                  = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  IS                           ip = a->row;
  const PetscInt               *rip;
  PetscBool                    perm_identity;
  PetscInt                     n = A->rmap->n;

  PetscFunctionBegin;
  ierr = MatSeqAIJCUSPARSEBuildICCTriMatrices(A);CHKERRQ(ierr);
  cusparseTriFactors->workVector = new THRUSTARRAY(n);
  cusparseTriFactors->nnz=(a->nz-n)*2 + n;

  /*lower triangular indices */
  ierr = ISGetIndices(ip,&rip);CHKERRQ(ierr);
  ierr = ISIdentity(ip,&perm_identity);CHKERRQ(ierr);
  if (!perm_identity) {
    cusparseTriFactors->rpermIndices = new THRUSTINTARRAY(n);
    cusparseTriFactors->rpermIndices->assign(rip, rip+n);
    cusparseTriFactors->cpermIndices = new THRUSTINTARRAY(n);
    cusparseTriFactors->cpermIndices->assign(rip, rip+n);
  }
  ierr = ISRestoreIndices(ip,&rip);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatLUFactorNumeric_SeqAIJCUSPARSE(Mat B,Mat A,const MatFactorInfo *info)
{
  Mat_SeqAIJ     *b = (Mat_SeqAIJ*)B->data;
  IS             isrow = b->row,iscol = b->col;
  PetscBool      row_identity,col_identity;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatLUFactorNumeric_SeqAIJ(B,A,info);CHKERRQ(ierr);
  /* determine which version of MatSolve needs to be used. */
  ierr = ISIdentity(isrow,&row_identity);CHKERRQ(ierr);
  ierr = ISIdentity(iscol,&col_identity);CHKERRQ(ierr);
  if (row_identity && col_identity) {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE_NaturalOrdering;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering;
  } else {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE;
  }

  /* get the triangular factors */
  ierr = MatSeqAIJCUSPARSEILUAnalysisAndCopyToGPU(B);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatCholeskyFactorNumeric_SeqAIJCUSPARSE(Mat B,Mat A,const MatFactorInfo *info)
{
  Mat_SeqAIJ     *b = (Mat_SeqAIJ*)B->data;
  IS             ip = b->row;
  PetscBool      perm_identity;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCholeskyFactorNumeric_SeqAIJ(B,A,info);CHKERRQ(ierr);

  /* determine which version of MatSolve needs to be used. */
  ierr = ISIdentity(ip,&perm_identity);CHKERRQ(ierr);
  if (perm_identity) {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE_NaturalOrdering;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering;
  } else {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE;
  }

  /* get the triangular factors */
  ierr = MatSeqAIJCUSPARSEICCAnalysisAndCopyToGPU(B);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEAnalyzeTransposeForSolve(Mat A)
{
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  hipsparseStatus_t                  stat;
  hipsparseIndexBase_t               indexBase;
  hipsparseMatrixType_t              matrixType;
  hipsparseFillMode_t                fillMode;
  hipsparseDiagType_t                diagType;

  PetscFunctionBegin;

  /*********************************************/
  /* Now the Transpose of the Lower Tri Factor */
  /*********************************************/

  /* allocate space for the transpose of the lower triangular factor */
  loTriFactorT = new Mat_SeqAIJCUSPARSETriFactorStruct;

  /* set the matrix descriptors of the lower triangular factor */
  matrixType = hipsparseGetMatType(loTriFactor->descr);
  indexBase = hipsparseGetMatIndexBase(loTriFactor->descr);
  fillMode = hipsparseGetMatFillMode(loTriFactor->descr)==HIPSPARSE_FILL_MODE_UPPER ?
    HIPSPARSE_FILL_MODE_LOWER : HIPSPARSE_FILL_MODE_UPPER;
  diagType = hipsparseGetMatDiagType(loTriFactor->descr);

  /* Create the matrix description */
  stat = hipsparseCreateMatDescr(&loTriFactorT->descr);CHKERRCUDA(stat);
  stat = hipsparseSetMatIndexBase(loTriFactorT->descr, indexBase);CHKERRCUDA(stat);
  stat = hipsparseSetMatType(loTriFactorT->descr, matrixType);CHKERRCUDA(stat);
  stat = hipsparseSetMatFillMode(loTriFactorT->descr, fillMode);CHKERRCUDA(stat);
  stat = hipsparseSetMatDiagType(loTriFactorT->descr, diagType);CHKERRCUDA(stat);

  /* Create the solve analysis information */
  stat = cusparseCreateSolveAnalysisInfo(&loTriFactorT->solveInfo);CHKERRCUDA(stat);

  /* set the operation */
  loTriFactorT->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  /* allocate GPU space for the CSC of the lower triangular factor*/
  loTriFactorT->csrMat = new CsrMatrix;
  loTriFactorT->csrMat->num_rows = loTriFactor->csrMat->num_rows;
  loTriFactorT->csrMat->num_cols = loTriFactor->csrMat->num_cols;
  loTriFactorT->csrMat->num_entries = loTriFactor->csrMat->num_entries;
  loTriFactorT->csrMat->row_offsets = new THRUSTINTARRAY32(loTriFactor->csrMat->num_rows+1);
  loTriFactorT->csrMat->column_indices = new THRUSTINTARRAY32(loTriFactor->csrMat->num_entries);
  loTriFactorT->csrMat->values = new THRUSTARRAY(loTriFactor->csrMat->num_entries);

  /* compute the transpose of the lower triangular factor, i.e. the CSC */
  stat = cusparse_csr2csc(cusparseTriFactors->handle, loTriFactor->csrMat->num_rows,
                          loTriFactor->csrMat->num_cols, loTriFactor->csrMat->num_entries,
                          loTriFactor->csrMat->values->data().get(),
                          loTriFactor->csrMat->row_offsets->data().get(),
                          loTriFactor->csrMat->column_indices->data().get(),
                          loTriFactorT->csrMat->values->data().get(),
                          loTriFactorT->csrMat->column_indices->data().get(),
                          loTriFactorT->csrMat->row_offsets->data().get(),
                          HIPSPARSE_ACTION_NUMERIC, indexBase);CHKERRCUDA(stat);

  /* perform the solve analysis on the transposed matrix */
  stat = cusparse_analysis(cusparseTriFactors->handle, loTriFactorT->solveOp,
                           loTriFactorT->csrMat->num_rows, loTriFactorT->csrMat->num_entries,
                           loTriFactorT->descr, loTriFactorT->csrMat->values->data().get(),
                           loTriFactorT->csrMat->row_offsets->data().get(), loTriFactorT->csrMat->column_indices->data().get(),
                           loTriFactorT->solveInfo);CHKERRCUDA(stat);

  /* assign the pointer. Is this really necessary? */
  ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->loTriFactorPtrTranspose = loTriFactorT;

  /*********************************************/
  /* Now the Transpose of the Upper Tri Factor */
  /*********************************************/

  /* allocate space for the transpose of the upper triangular factor */
  upTriFactorT = new Mat_SeqAIJCUSPARSETriFactorStruct;

  /* set the matrix descriptors of the upper triangular factor */
  matrixType = hipsparseGetMatType(upTriFactor->descr);
  indexBase = hipsparseGetMatIndexBase(upTriFactor->descr);
  fillMode = hipsparseGetMatFillMode(upTriFactor->descr)==HIPSPARSE_FILL_MODE_UPPER ?
    HIPSPARSE_FILL_MODE_LOWER : HIPSPARSE_FILL_MODE_UPPER;
  diagType = hipsparseGetMatDiagType(upTriFactor->descr);

  /* Create the matrix description */
  stat = hipsparseCreateMatDescr(&upTriFactorT->descr);CHKERRCUDA(stat);
  stat = hipsparseSetMatIndexBase(upTriFactorT->descr, indexBase);CHKERRCUDA(stat);
  stat = hipsparseSetMatType(upTriFactorT->descr, matrixType);CHKERRCUDA(stat);
  stat = hipsparseSetMatFillMode(upTriFactorT->descr, fillMode);CHKERRCUDA(stat);
  stat = hipsparseSetMatDiagType(upTriFactorT->descr, diagType);CHKERRCUDA(stat);

  /* Create the solve analysis information */
  stat = cusparseCreateSolveAnalysisInfo(&upTriFactorT->solveInfo);CHKERRCUDA(stat);

  /* set the operation */
  upTriFactorT->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  /* allocate GPU space for the CSC of the upper triangular factor*/
  upTriFactorT->csrMat = new CsrMatrix;
  upTriFactorT->csrMat->num_rows = upTriFactor->csrMat->num_rows;
  upTriFactorT->csrMat->num_cols = upTriFactor->csrMat->num_cols;
  upTriFactorT->csrMat->num_entries = upTriFactor->csrMat->num_entries;
  upTriFactorT->csrMat->row_offsets = new THRUSTINTARRAY32(upTriFactor->csrMat->num_rows+1);
  upTriFactorT->csrMat->column_indices = new THRUSTINTARRAY32(upTriFactor->csrMat->num_entries);
  upTriFactorT->csrMat->values = new THRUSTARRAY(upTriFactor->csrMat->num_entries);

  /* compute the transpose of the upper triangular factor, i.e. the CSC */
  stat = cusparse_csr2csc(cusparseTriFactors->handle, upTriFactor->csrMat->num_rows,
                          upTriFactor->csrMat->num_cols, upTriFactor->csrMat->num_entries,
                          upTriFactor->csrMat->values->data().get(),
                          upTriFactor->csrMat->row_offsets->data().get(),
                          upTriFactor->csrMat->column_indices->data().get(),
                          upTriFactorT->csrMat->values->data().get(),
                          upTriFactorT->csrMat->column_indices->data().get(),
                          upTriFactorT->csrMat->row_offsets->data().get(),
                          HIPSPARSE_ACTION_NUMERIC, indexBase);CHKERRCUDA(stat);

  /* perform the solve analysis on the transposed matrix */
  stat = cusparse_analysis(cusparseTriFactors->handle, upTriFactorT->solveOp,
                           upTriFactorT->csrMat->num_rows, upTriFactorT->csrMat->num_entries,
                           upTriFactorT->descr, upTriFactorT->csrMat->values->data().get(),
                           upTriFactorT->csrMat->row_offsets->data().get(), upTriFactorT->csrMat->column_indices->data().get(),
                           upTriFactorT->solveInfo);CHKERRCUDA(stat);

  /* assign the pointer. Is this really necessary? */
  ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->upTriFactorPtrTranspose = upTriFactorT;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEGenerateTransposeForMult(Mat A)
{
  Mat_SeqAIJCUSPARSE           *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct *matstruct = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->mat;
  Mat_SeqAIJCUSPARSEMultStruct *matstructT = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->matTranspose;
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  hipsparseStatus_t             stat;
  hipsparseIndexBase_t          indexBase;
  hipError_t                  err;

  PetscFunctionBegin;

  /* allocate space for the triangular factor information */
  matstructT = new Mat_SeqAIJCUSPARSEMultStruct;
  stat = hipsparseCreateMatDescr(&matstructT->descr);CHKERRCUDA(stat);
  indexBase = hipsparseGetMatIndexBase(matstruct->descr);
  stat = hipsparseSetMatIndexBase(matstructT->descr, indexBase);CHKERRCUDA(stat);
  stat = hipsparseSetMatType(matstructT->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);CHKERRCUDA(stat);

  /* set alpha and beta */
  err = hipMalloc((void **)&(matstructT->alpha),    sizeof(PetscScalar));CHKERRCUDA(err);
  err = hipMalloc((void **)&(matstructT->beta_zero),sizeof(PetscScalar));CHKERRCUDA(err);
  err = hipMalloc((void **)&(matstructT->beta_one), sizeof(PetscScalar));CHKERRCUDA(err);
  err = hipMemcpy(matstructT->alpha,    &PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);
  err = hipMemcpy(matstructT->beta_zero,&PETSC_CUSPARSE_ZERO,sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);
  err = hipMemcpy(matstructT->beta_one, &PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);
  stat = hipsparseSetPointerMode(cusparsestruct->handle, HIPSPARSE_POINTER_MODE_DEVICE);CHKERRCUDA(stat);

  if (cusparsestruct->format==MAT_CUSPARSE_CSR) {
    CsrMatrix *matrix = (CsrMatrix*)matstruct->mat;
    CsrMatrix *matrixT= new CsrMatrix;
    matrixT->num_rows = A->cmap->n;
    matrixT->num_cols = A->rmap->n;
    matrixT->num_entries = a->nz;
    matrixT->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
    matrixT->column_indices = new THRUSTINTARRAY32(a->nz);
    matrixT->values = new THRUSTARRAY(a->nz);

    /* compute the transpose of the upper triangular factor, i.e. the CSC */
    indexBase = hipsparseGetMatIndexBase(matstruct->descr);
    stat = cusparse_csr2csc(cusparsestruct->handle, matrix->num_rows,
                            matrix->num_cols, matrix->num_entries,
                            matrix->values->data().get(),
                            matrix->row_offsets->data().get(),
                            matrix->column_indices->data().get(),
                            matrixT->values->data().get(),
                            matrixT->column_indices->data().get(),
                            matrixT->row_offsets->data().get(),
                            HIPSPARSE_ACTION_NUMERIC, indexBase);CHKERRCUDA(stat);

    /* assign the pointer */
    matstructT->mat = matrixT;

  } else if (cusparsestruct->format==MAT_CUSPARSE_ELL || cusparsestruct->format==MAT_CUSPARSE_HYB) {
#if CUDA_VERSION>=5000
    /* First convert HYB to CSR */
    CsrMatrix *temp= new CsrMatrix;
    temp->num_rows = A->rmap->n;
    temp->num_cols = A->cmap->n;
    temp->num_entries = a->nz;
    temp->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
    temp->column_indices = new THRUSTINTARRAY32(a->nz);
    temp->values = new THRUSTARRAY(a->nz);


    stat = cusparse_hyb2csr(cusparsestruct->handle,
                            matstruct->descr, (hipsparseHybMat_t)matstruct->mat,
                            temp->values->data().get(),
                            temp->row_offsets->data().get(),
                            temp->column_indices->data().get());CHKERRCUDA(stat);

    /* Next, convert CSR to CSC (i.e. the matrix transpose) */
    CsrMatrix *tempT= new CsrMatrix;
    tempT->num_rows = A->rmap->n;
    tempT->num_cols = A->cmap->n;
    tempT->num_entries = a->nz;
    tempT->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
    tempT->column_indices = new THRUSTINTARRAY32(a->nz);
    tempT->values = new THRUSTARRAY(a->nz);

    stat = cusparse_csr2csc(cusparsestruct->handle, temp->num_rows,
                            temp->num_cols, temp->num_entries,
                            temp->values->data().get(),
                            temp->row_offsets->data().get(),
                            temp->column_indices->data().get(),
                            tempT->values->data().get(),
                            tempT->column_indices->data().get(),
                            tempT->row_offsets->data().get(),
                            HIPSPARSE_ACTION_NUMERIC, indexBase);CHKERRCUDA(stat);

    /* Last, convert CSC to HYB */
    hipsparseHybMat_t hybMat;
    stat = hipsparseCreateHybMat(&hybMat);CHKERRCUDA(stat);
    hipsparseHybPartition_t partition = cusparsestruct->format==MAT_CUSPARSE_ELL ?
      HIPSPARSE_HYB_PARTITION_MAX : HIPSPARSE_HYB_PARTITION_AUTO;
    stat = cusparse_csr2hyb(cusparsestruct->handle, A->rmap->n, A->cmap->n,
                            matstructT->descr, tempT->values->data().get(),
                            tempT->row_offsets->data().get(),
                            tempT->column_indices->data().get(),
                            hybMat, 0, partition);CHKERRCUDA(stat);

    /* assign the pointer */
    matstructT->mat = hybMat;

    /* delete temporaries */
    if (tempT) {
      if (tempT->values) delete (THRUSTARRAY*) tempT->values;
      if (tempT->column_indices) delete (THRUSTINTARRAY32*) tempT->column_indices;
      if (tempT->row_offsets) delete (THRUSTINTARRAY32*) tempT->row_offsets;
      delete (CsrMatrix*) tempT;
    }
    if (temp) {
      if (temp->values) delete (THRUSTARRAY*) temp->values;
      if (temp->column_indices) delete (THRUSTINTARRAY32*) temp->column_indices;
      if (temp->row_offsets) delete (THRUSTINTARRAY32*) temp->row_offsets;
      delete (CsrMatrix*) temp;
    }
#else
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"ELL (Ellpack) and HYB (Hybrid) storage format for the Matrix Transpose (in MatMultTranspose) require CUDA 5.0 or later.");
#endif
  }
  /* assign the compressed row indices */
  matstructT->cprowIndices = new THRUSTINTARRAY;
  matstructT->cprowIndices->resize(A->cmap->n);
  thrust::sequence(matstructT->cprowIndices->begin(), matstructT->cprowIndices->end());

  /* assign the pointer */
  ((Mat_SeqAIJCUSPARSE*)A->spptr)->matTranspose = matstructT;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE(Mat A,Vec bb,Vec xx)
{
  PetscInt                              n = xx->map->n;
  const PetscScalar                     *barray;
  PetscScalar                           *xarray;
  thrust::device_ptr<const PetscScalar> bGPU;
  thrust::device_ptr<PetscScalar>       xGPU;
  hipsparseStatus_t                      stat;
  Mat_SeqAIJCUSPARSETriFactors          *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct     *loTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
  Mat_SeqAIJCUSPARSETriFactorStruct     *upTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  THRUSTARRAY                           *tempGPU = (THRUSTARRAY*)cusparseTriFactors->workVector;
  PetscErrorCode                        ierr;

  PetscFunctionBegin;
  /* Analyze the matrix and create the transpose ... on the fly */
  if (!loTriFactorT && !upTriFactorT) {
    ierr = MatSeqAIJCUSPARSEAnalyzeTransposeForSolve(A);CHKERRQ(ierr);
    loTriFactorT       = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
    upTriFactorT       = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  }

  /* Get the GPU pointers */
  ierr = VecCUDAGetArrayWrite(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(bb,&barray);CHKERRQ(ierr);
  xGPU = thrust::device_pointer_cast(xarray);
  bGPU = thrust::device_pointer_cast(barray);

  /* First, reorder with the row permutation */
  thrust::copy(thrust::make_permutation_iterator(bGPU, cusparseTriFactors->rpermIndices->begin()),
               thrust::make_permutation_iterator(bGPU+n, cusparseTriFactors->rpermIndices->end()),
               xGPU);

  /* First, solve U */
  stat = cusparse_solve(cusparseTriFactors->handle, upTriFactorT->solveOp,
                        upTriFactorT->csrMat->num_rows, &PETSC_CUSPARSE_ONE, upTriFactorT->descr,
                        upTriFactorT->csrMat->values->data().get(),
                        upTriFactorT->csrMat->row_offsets->data().get(),
                        upTriFactorT->csrMat->column_indices->data().get(),
                        upTriFactorT->solveInfo,
                        xarray, tempGPU->data().get());CHKERRCUDA(stat);

  /* Then, solve L */
  stat = cusparse_solve(cusparseTriFactors->handle, loTriFactorT->solveOp,
                        loTriFactorT->csrMat->num_rows, &PETSC_CUSPARSE_ONE, loTriFactorT->descr,
                        loTriFactorT->csrMat->values->data().get(),
                        loTriFactorT->csrMat->row_offsets->data().get(),
                        loTriFactorT->csrMat->column_indices->data().get(),
                        loTriFactorT->solveInfo,
                        tempGPU->data().get(), xarray);CHKERRCUDA(stat);

  /* Last, copy the solution, xGPU, into a temporary with the column permutation ... can't be done in place. */
  thrust::copy(thrust::make_permutation_iterator(xGPU, cusparseTriFactors->cpermIndices->begin()),
               thrust::make_permutation_iterator(xGPU+n, cusparseTriFactors->cpermIndices->end()),
               tempGPU->begin());

  /* Copy the temporary to the full solution. */
  thrust::copy(tempGPU->begin(), tempGPU->end(), xGPU);

  /* restore */
  ierr = VecCUDARestoreArrayRead(bb,&barray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayWrite(xx,&xarray);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUDA(ierr);

  ierr = PetscLogFlops(2.0*cusparseTriFactors->nnz - A->cmap->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering(Mat A,Vec bb,Vec xx)
{
  const PetscScalar                 *barray;
  PetscScalar                       *xarray;
  hipsparseStatus_t                  stat;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  THRUSTARRAY                       *tempGPU = (THRUSTARRAY*)cusparseTriFactors->workVector;
  PetscErrorCode                    ierr;

  PetscFunctionBegin;
  /* Analyze the matrix and create the transpose ... on the fly */
  if (!loTriFactorT && !upTriFactorT) {
    ierr = MatSeqAIJCUSPARSEAnalyzeTransposeForSolve(A);CHKERRQ(ierr);
    loTriFactorT       = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
    upTriFactorT       = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  }

  /* Get the GPU pointers */
  ierr = VecCUDAGetArrayWrite(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(bb,&barray);CHKERRQ(ierr);

  /* First, solve U */
  stat = cusparse_solve(cusparseTriFactors->handle, upTriFactorT->solveOp,
                        upTriFactorT->csrMat->num_rows, &PETSC_CUSPARSE_ONE, upTriFactorT->descr,
                        upTriFactorT->csrMat->values->data().get(),
                        upTriFactorT->csrMat->row_offsets->data().get(),
                        upTriFactorT->csrMat->column_indices->data().get(),
                        upTriFactorT->solveInfo,
                        barray, tempGPU->data().get());CHKERRCUDA(stat);

  /* Then, solve L */
  stat = cusparse_solve(cusparseTriFactors->handle, loTriFactorT->solveOp,
                        loTriFactorT->csrMat->num_rows, &PETSC_CUSPARSE_ONE, loTriFactorT->descr,
                        loTriFactorT->csrMat->values->data().get(),
                        loTriFactorT->csrMat->row_offsets->data().get(),
                        loTriFactorT->csrMat->column_indices->data().get(),
                        loTriFactorT->solveInfo,
                        tempGPU->data().get(), xarray);CHKERRCUDA(stat);

  /* restore */
  ierr = VecCUDARestoreArrayRead(bb,&barray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayWrite(xx,&xarray);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  ierr = PetscLogFlops(2.0*cusparseTriFactors->nnz - A->cmap->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqAIJCUSPARSE(Mat A,Vec bb,Vec xx)
{
  const PetscScalar                     *barray;
  PetscScalar                           *xarray;
  thrust::device_ptr<const PetscScalar> bGPU;
  thrust::device_ptr<PetscScalar>       xGPU;
  hipsparseStatus_t                      stat;
  Mat_SeqAIJCUSPARSETriFactors          *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct     *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct     *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  THRUSTARRAY                           *tempGPU = (THRUSTARRAY*)cusparseTriFactors->workVector;
  PetscErrorCode                        ierr;

  PetscFunctionBegin;

  /* Get the GPU pointers */
  ierr = VecCUDAGetArrayWrite(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(bb,&barray);CHKERRQ(ierr);
  xGPU = thrust::device_pointer_cast(xarray);
  bGPU = thrust::device_pointer_cast(barray);

  /* First, reorder with the row permutation */
  thrust::copy(thrust::make_permutation_iterator(bGPU, cusparseTriFactors->rpermIndices->begin()),
               thrust::make_permutation_iterator(bGPU, cusparseTriFactors->rpermIndices->end()),
               xGPU);

  /* Next, solve L */
  stat = cusparse_solve(cusparseTriFactors->handle, loTriFactor->solveOp,
                        loTriFactor->csrMat->num_rows, &PETSC_CUSPARSE_ONE, loTriFactor->descr,
                        loTriFactor->csrMat->values->data().get(),
                        loTriFactor->csrMat->row_offsets->data().get(),
                        loTriFactor->csrMat->column_indices->data().get(),
                        loTriFactor->solveInfo,
                        xarray, tempGPU->data().get());CHKERRCUDA(stat);

  /* Then, solve U */
  stat = cusparse_solve(cusparseTriFactors->handle, upTriFactor->solveOp,
                        upTriFactor->csrMat->num_rows, &PETSC_CUSPARSE_ONE, upTriFactor->descr,
                        upTriFactor->csrMat->values->data().get(),
                        upTriFactor->csrMat->row_offsets->data().get(),
                        upTriFactor->csrMat->column_indices->data().get(),
                        upTriFactor->solveInfo,
                        tempGPU->data().get(), xarray);CHKERRCUDA(stat);

  /* Last, copy the solution, xGPU, into a temporary with the column permutation ... can't be done in place. */
  thrust::copy(thrust::make_permutation_iterator(xGPU, cusparseTriFactors->cpermIndices->begin()),
               thrust::make_permutation_iterator(xGPU, cusparseTriFactors->cpermIndices->end()),
               tempGPU->begin());

  /* Copy the temporary to the full solution. */
  thrust::copy(tempGPU->begin(), tempGPU->end(), xGPU);

  ierr = VecCUDARestoreArrayRead(bb,&barray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayWrite(xx,&xarray);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  ierr = PetscLogFlops(2.0*cusparseTriFactors->nnz - A->cmap->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqAIJCUSPARSE_NaturalOrdering(Mat A,Vec bb,Vec xx)
{
  const PetscScalar                 *barray;
  PetscScalar                       *xarray;
  hipsparseStatus_t                  stat;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  THRUSTARRAY                       *tempGPU = (THRUSTARRAY*)cusparseTriFactors->workVector;
  PetscErrorCode                    ierr;

  PetscFunctionBegin;
  /* Get the GPU pointers */
  ierr = VecCUDAGetArrayWrite(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(bb,&barray);CHKERRQ(ierr);

  /* First, solve L */
  stat = cusparse_solve(cusparseTriFactors->handle, loTriFactor->solveOp,
                        loTriFactor->csrMat->num_rows, &PETSC_CUSPARSE_ONE, loTriFactor->descr,
                        loTriFactor->csrMat->values->data().get(),
                        loTriFactor->csrMat->row_offsets->data().get(),
                        loTriFactor->csrMat->column_indices->data().get(),
                        loTriFactor->solveInfo,
                        barray, tempGPU->data().get());CHKERRCUDA(stat);

  /* Next, solve U */
  stat = cusparse_solve(cusparseTriFactors->handle, upTriFactor->solveOp,
                        upTriFactor->csrMat->num_rows, &PETSC_CUSPARSE_ONE, upTriFactor->descr,
                        upTriFactor->csrMat->values->data().get(),
                        upTriFactor->csrMat->row_offsets->data().get(),
                        upTriFactor->csrMat->column_indices->data().get(),
                        upTriFactor->solveInfo,
                        tempGPU->data().get(), xarray);CHKERRCUDA(stat);

  ierr = VecCUDARestoreArrayRead(bb,&barray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayWrite(xx,&xarray);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  ierr = PetscLogFlops(2.0*cusparseTriFactors->nnz - A->cmap->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSECopyToGPU(Mat A)
{

  Mat_SeqAIJCUSPARSE           *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct *matstruct = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->mat;
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  PetscInt                     m = A->rmap->n,*ii,*ridx;
  PetscErrorCode               ierr;
  hipsparseStatus_t             stat;
  hipError_t                  err;

  PetscFunctionBegin;
  if (A->valid_GPU_matrix == PETSC_OFFLOAD_UNALLOCATED || A->valid_GPU_matrix == PETSC_OFFLOAD_CPU) {
    ierr = PetscLogEventBegin(MAT_CUSPARSECopyToGPU,A,0,0,0);CHKERRQ(ierr);
    if (A->assembled && A->nonzerostate == cusparsestruct->nonzerostate && cusparsestruct->format == MAT_CUSPARSE_CSR) {
      CsrMatrix *matrix = (CsrMatrix*)matstruct->mat;
      /* copy values only */
      matrix->values->assign(a->a, a->a+a->nz);
    } else {
      MatSeqAIJCUSPARSEMultStruct_Destroy(&matstruct,cusparsestruct->format);
      try {
        cusparsestruct->nonzerorow=0;
        for (int j = 0; j<m; j++) cusparsestruct->nonzerorow += ((a->i[j+1]-a->i[j])>0);

        if (a->compressedrow.use) {
          m    = a->compressedrow.nrows;
          ii   = a->compressedrow.i;
          ridx = a->compressedrow.rindex;
        } else {
          /* Forcing compressed row on the GPU */
          int k=0;
          ierr = PetscMalloc1(cusparsestruct->nonzerorow+1, &ii);CHKERRQ(ierr);
          ierr = PetscMalloc1(cusparsestruct->nonzerorow, &ridx);CHKERRQ(ierr);
          ii[0]=0;
          for (int j = 0; j<m; j++) {
            if ((a->i[j+1]-a->i[j])>0) {
              ii[k]  = a->i[j];
              ridx[k]= j;
              k++;
            }
          }
          ii[cusparsestruct->nonzerorow] = a->nz;
          m = cusparsestruct->nonzerorow;
        }

        /* allocate space for the triangular factor information */
        matstruct = new Mat_SeqAIJCUSPARSEMultStruct;
        stat = hipsparseCreateMatDescr(&matstruct->descr);CHKERRCUDA(stat);
        stat = hipsparseSetMatIndexBase(matstruct->descr, HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUDA(stat);
        stat = hipsparseSetMatType(matstruct->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);CHKERRCUDA(stat);

        err = hipMalloc((void **)&(matstruct->alpha),    sizeof(PetscScalar));CHKERRCUDA(err);
        err = hipMalloc((void **)&(matstruct->beta_zero),sizeof(PetscScalar));CHKERRCUDA(err);
        err = hipMalloc((void **)&(matstruct->beta_one), sizeof(PetscScalar));CHKERRCUDA(err);
        err = hipMemcpy(matstruct->alpha,    &PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);
        err = hipMemcpy(matstruct->beta_zero,&PETSC_CUSPARSE_ZERO,sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);
        err = hipMemcpy(matstruct->beta_one, &PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(err);
        stat = hipsparseSetPointerMode(cusparsestruct->handle, HIPSPARSE_POINTER_MODE_DEVICE);CHKERRCUDA(stat);

        /* Build a hybrid/ellpack matrix if this option is chosen for the storage */
        if (cusparsestruct->format==MAT_CUSPARSE_CSR) {
          /* set the matrix */
          CsrMatrix *matrix= new CsrMatrix;
          matrix->num_rows = m;
          matrix->num_cols = A->cmap->n;
          matrix->num_entries = a->nz;
          matrix->row_offsets = new THRUSTINTARRAY32(m+1);
          matrix->row_offsets->assign(ii, ii + m+1);

          matrix->column_indices = new THRUSTINTARRAY32(a->nz);
          matrix->column_indices->assign(a->j, a->j+a->nz);

          matrix->values = new THRUSTARRAY(a->nz);
          matrix->values->assign(a->a, a->a+a->nz);

          /* assign the pointer */
          matstruct->mat = matrix;

        } else if (cusparsestruct->format==MAT_CUSPARSE_ELL || cusparsestruct->format==MAT_CUSPARSE_HYB) {
#if CUDA_VERSION>=4020
          CsrMatrix *matrix= new CsrMatrix;
          matrix->num_rows = m;
          matrix->num_cols = A->cmap->n;
          matrix->num_entries = a->nz;
          matrix->row_offsets = new THRUSTINTARRAY32(m+1);
          matrix->row_offsets->assign(ii, ii + m+1);

          matrix->column_indices = new THRUSTINTARRAY32(a->nz);
          matrix->column_indices->assign(a->j, a->j+a->nz);

          matrix->values = new THRUSTARRAY(a->nz);
          matrix->values->assign(a->a, a->a+a->nz);

          hipsparseHybMat_t hybMat;
          stat = hipsparseCreateHybMat(&hybMat);CHKERRCUDA(stat);
          hipsparseHybPartition_t partition = cusparsestruct->format==MAT_CUSPARSE_ELL ?
            HIPSPARSE_HYB_PARTITION_MAX : HIPSPARSE_HYB_PARTITION_AUTO;
          stat = cusparse_csr2hyb(cusparsestruct->handle, matrix->num_rows, matrix->num_cols,
              matstruct->descr, matrix->values->data().get(),
              matrix->row_offsets->data().get(),
              matrix->column_indices->data().get(),
              hybMat, 0, partition);CHKERRCUDA(stat);
          /* assign the pointer */
          matstruct->mat = hybMat;

          if (matrix) {
            if (matrix->values) delete (THRUSTARRAY*)matrix->values;
            if (matrix->column_indices) delete (THRUSTINTARRAY32*)matrix->column_indices;
            if (matrix->row_offsets) delete (THRUSTINTARRAY32*)matrix->row_offsets;
            delete (CsrMatrix*)matrix;
          }
#endif
        }

        /* assign the compressed row indices */
        matstruct->cprowIndices = new THRUSTINTARRAY(m);
        matstruct->cprowIndices->assign(ridx,ridx+m);

        /* assign the pointer */
        cusparsestruct->mat = matstruct;

        if (!a->compressedrow.use) {
          ierr = PetscFree(ii);CHKERRQ(ierr);
          ierr = PetscFree(ridx);CHKERRQ(ierr);
        }
        cusparsestruct->workVector = new THRUSTARRAY(m);
      } catch(char *ex) {
        SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
      }
      cusparsestruct->nonzerostate = A->nonzerostate;
    }
    ierr = WaitForGPU();CHKERRCUDA(ierr);
    A->valid_GPU_matrix = PETSC_OFFLOAD_BOTH;
    ierr = PetscLogEventEnd(MAT_CUSPARSECopyToGPU,A,0,0,0);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

struct VecCUDAPlusEquals
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<1>(t) = thrust::get<1>(t) + thrust::get<0>(t);
  }
};

static PetscErrorCode MatMult_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAdd_SeqAIJCUSPARSE(A,xx,NULL,yy);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMultTranspose_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSE           *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct *matstructT;
  const PetscScalar            *xarray;
  PetscScalar                  *yarray;
  PetscErrorCode               ierr;
  hipsparseStatus_t             stat;

  PetscFunctionBegin;
  /* The line below is necessary due to the operations that modify the matrix on the CPU (axpy, scale, etc) */
  ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
  matstructT = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->matTranspose;
  if (!matstructT) {
    ierr = MatSeqAIJCUSPARSEGenerateTransposeForMult(A);CHKERRQ(ierr);
    matstructT = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->matTranspose;
  }
  ierr = VecCUDAGetArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecSet(yy,0);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayWrite(yy,&yarray);CHKERRQ(ierr);

  if (cusparsestruct->format==MAT_CUSPARSE_CSR) {
    CsrMatrix *mat = (CsrMatrix*)matstructT->mat;
    stat = cusparse_csr_spmv(cusparsestruct->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             mat->num_rows, mat->num_cols,
                             mat->num_entries, matstructT->alpha, matstructT->descr,
                             mat->values->data().get(), mat->row_offsets->data().get(),
                             mat->column_indices->data().get(), xarray, matstructT->beta_zero,
                             yarray);CHKERRCUDA(stat);
  } else {
#if CUDA_VERSION>=4020
    hipsparseHybMat_t hybMat = (hipsparseHybMat_t)matstructT->mat;
    stat = cusparse_hyb_spmv(cusparsestruct->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             matstructT->alpha, matstructT->descr, hybMat,
                             xarray, matstructT->beta_zero,
                             yarray);CHKERRCUDA(stat);
#endif
  }
  ierr = VecCUDARestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayWrite(yy,&yarray);CHKERRQ(ierr);
  if (!cusparsestruct->stream) {
    ierr = WaitForGPU();CHKERRCUDA(ierr);
  }
  ierr = PetscLogFlops(2.0*a->nz - cusparsestruct->nonzerorow);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


static PetscErrorCode MatMultAdd_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy,Vec zz)
{
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSE           *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct *matstruct;
  const PetscScalar            *xarray;
  PetscScalar                  *zarray,*dptr,*beta;
  PetscErrorCode               ierr;
  hipsparseStatus_t             stat;

  PetscFunctionBegin;
  /* The line below is necessary due to the operations that modify the matrix on the CPU (axpy, scale, etc) */
  ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
  matstruct = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->mat;
  try {
    ierr = VecCUDAGetArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUDAGetArrayReadWrite(zz,&zarray);CHKERRQ(ierr);
    dptr = cusparsestruct->workVector->size() == (thrust::detail::vector_base<PetscScalar, thrust::device_malloc_allocator<PetscScalar> >::size_type)(A->rmap->n) ? zarray : cusparsestruct->workVector->data().get();
    beta = (yy == zz && dptr == zarray) ? matstruct->beta_one : matstruct->beta_zero;

    /* csr_spmv is multiply add */
    if (cusparsestruct->format == MAT_CUSPARSE_CSR) {
      /* here we need to be careful to set the number of rows in the multiply to the
         number of compressed rows in the matrix ... which is equivalent to the
         size of the workVector */
      CsrMatrix *mat = (CsrMatrix*)matstruct->mat;
      stat = cusparse_csr_spmv(cusparsestruct->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               mat->num_rows, mat->num_cols,
                               mat->num_entries, matstruct->alpha, matstruct->descr,
                               mat->values->data().get(), mat->row_offsets->data().get(),
                               mat->column_indices->data().get(), xarray, beta,
                               dptr);CHKERRCUDA(stat);
    } else {
#if CUDA_VERSION>=4020
      hipsparseHybMat_t hybMat = (hipsparseHybMat_t)matstruct->mat;
      if (cusparsestruct->workVector->size()) {
        stat = cusparse_hyb_spmv(cusparsestruct->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 matstruct->alpha, matstruct->descr, hybMat,
                                 xarray, beta,
                                 dptr);CHKERRCUDA(stat);
      }
#endif
    }

    if (yy) {
      if (dptr != zarray) {
        ierr = VecCopy_SeqCUDA(yy,zz);CHKERRQ(ierr);
      } else if (zz != yy) {
        ierr = VecAXPY_SeqCUDA(zz,1.0,yy);CHKERRQ(ierr);
      }
    } else if (dptr != zarray) {
      ierr = VecSet(zz,0);CHKERRQ(ierr);
    }
    /* scatter the data from the temporary into the full vector with a += operation */
    if (dptr != zarray) {
      thrust::device_ptr<PetscScalar> zptr;

      zptr = thrust::device_pointer_cast(zarray);
      thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(cusparsestruct->workVector->begin(), thrust::make_permutation_iterator(zptr, matstruct->cprowIndices->begin()))),
                       thrust::make_zip_iterator(thrust::make_tuple(cusparsestruct->workVector->begin(), thrust::make_permutation_iterator(zptr, matstruct->cprowIndices->begin()))) + cusparsestruct->workVector->size(),
                       VecCUDAPlusEquals());
    }
    ierr = VecCUDARestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUDARestoreArrayReadWrite(zz,&zarray);CHKERRQ(ierr);
  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
  }
  if (!yy) { /* MatMult */
    if (!cusparsestruct->stream) {
      ierr = WaitForGPU();CHKERRCUDA(ierr);
    }
  }
  ierr = PetscLogFlops(2.0*a->nz);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMultTransposeAdd_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy,Vec zz)
{
  Mat_SeqAIJ                      *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSE              *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct    *matstructT;
  thrust::device_ptr<PetscScalar> zptr;
  const PetscScalar               *xarray;
  PetscScalar                     *zarray;
  PetscErrorCode                  ierr;
  hipsparseStatus_t                stat;

  PetscFunctionBegin;
  /* The line below is necessary due to the operations that modify the matrix on the CPU (axpy, scale, etc) */
  ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
  matstructT = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->matTranspose;
  if (!matstructT) {
    ierr = MatSeqAIJCUSPARSEGenerateTransposeForMult(A);CHKERRQ(ierr);
    matstructT = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->matTranspose;
  }

  try {
    ierr = VecCopy_SeqCUDA(yy,zz);CHKERRQ(ierr);
    ierr = VecCUDAGetArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUDAGetArrayReadWrite(zz,&zarray);CHKERRQ(ierr);
    zptr = thrust::device_pointer_cast(zarray);

    /* multiply add with matrix transpose */
    if (cusparsestruct->format==MAT_CUSPARSE_CSR) {
      CsrMatrix *mat = (CsrMatrix*)matstructT->mat;
      /* here we need to be careful to set the number of rows in the multiply to the
         number of compressed rows in the matrix ... which is equivalent to the
         size of the workVector */
      stat = cusparse_csr_spmv(cusparsestruct->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               mat->num_rows, mat->num_cols,
                               mat->num_entries, matstructT->alpha, matstructT->descr,
                               mat->values->data().get(), mat->row_offsets->data().get(),
                               mat->column_indices->data().get(), xarray, matstructT->beta_zero,
                               cusparsestruct->workVector->data().get());CHKERRCUDA(stat);
    } else {
#if CUDA_VERSION>=4020
      hipsparseHybMat_t hybMat = (hipsparseHybMat_t)matstructT->mat;
      if (cusparsestruct->workVector->size()) {
        stat = cusparse_hyb_spmv(cusparsestruct->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            matstructT->alpha, matstructT->descr, hybMat,
            xarray, matstructT->beta_zero,
            cusparsestruct->workVector->data().get());CHKERRCUDA(stat);
      }
#endif
    }

    /* scatter the data from the temporary into the full vector with a += operation */
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(cusparsestruct->workVector->begin(), thrust::make_permutation_iterator(zptr, matstructT->cprowIndices->begin()))),
        thrust::make_zip_iterator(thrust::make_tuple(cusparsestruct->workVector->begin(), thrust::make_permutation_iterator(zptr, matstructT->cprowIndices->begin()))) + A->cmap->n,
        VecCUDAPlusEquals());

    ierr = VecCUDARestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUDARestoreArrayReadWrite(zz,&zarray);CHKERRQ(ierr);

  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
  }
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  ierr = PetscLogFlops(2.0*a->nz);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatAssemblyEnd_SeqAIJCUSPARSE(Mat A,MatAssemblyType mode)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatAssemblyEnd_SeqAIJ(A,mode);CHKERRQ(ierr);
  if (A->factortype==MAT_FACTOR_NONE) {
    ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
  }
  if (mode == MAT_FLUSH_ASSEMBLY) PetscFunctionReturn(0);
  A->ops->mult             = MatMult_SeqAIJCUSPARSE;
  A->ops->multadd          = MatMultAdd_SeqAIJCUSPARSE;
  A->ops->multtranspose    = MatMultTranspose_SeqAIJCUSPARSE;
  A->ops->multtransposeadd = MatMultTransposeAdd_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

/* --------------------------------------------------------------------------------*/
/*@
   MatCreateSeqAIJCUSPARSE - Creates a sparse matrix in AIJ (compressed row) format
   (the default parallel PETSc format). This matrix will ultimately pushed down
   to NVidia GPUs and use the CUSPARSE library for calculations. For good matrix
   assembly performance the user should preallocate the matrix storage by setting
   the parameter nz (or the array nnz).  By setting these parameters accurately,
   performance during matrix assembly can be increased by more than a factor of 50.

   Collective on MPI_Comm

   Input Parameters:
+  comm - MPI communicator, set to PETSC_COMM_SELF
.  m - number of rows
.  n - number of columns
.  nz - number of nonzeros per row (same for all rows)
-  nnz - array containing the number of nonzeros in the various rows
         (possibly different for each row) or NULL

   Output Parameter:
.  A - the matrix

   It is recommended that one use the MatCreate(), MatSetType() and/or MatSetFromOptions(),
   MatXXXXSetPreallocation() paradgm instead of this routine directly.
   [MatXXXXSetPreallocation() is, for example, MatSeqAIJSetPreallocation]

   Notes:
   If nnz is given then nz is ignored

   The AIJ format (also called the Yale sparse matrix format or
   compressed row storage), is fully compatible with standard Fortran 77
   storage.  That is, the stored row and column indices can begin at
   either one (as in Fortran) or zero.  See the users' manual for details.

   Specify the preallocated storage with either nz or nnz (not both).
   Set nz=PETSC_DEFAULT and nnz=NULL for PETSc to control dynamic memory
   allocation.  For large problems you MUST preallocate memory or you
   will get TERRIBLE performance, see the users' manual chapter on matrices.

   By default, this format uses inodes (identical nodes) when possible, to
   improve numerical efficiency of matrix-vector products and solves. We
   search for consecutive rows with the same nonzero structure, thereby
   reusing matrix information to achieve increased efficiency.

   Level: intermediate

.seealso: MatCreate(), MatCreateAIJ(), MatSetValues(), MatSeqAIJSetColumnIndices(), MatCreateSeqAIJWithArrays(), MatCreateAIJ(), MATSEQAIJCUSPARSE, MATAIJCUSPARSE
@*/
PetscErrorCode  MatCreateSeqAIJCUSPARSE(MPI_Comm comm,PetscInt m,PetscInt n,PetscInt nz,const PetscInt nnz[],Mat *A)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCreate(comm,A);CHKERRQ(ierr);
  ierr = MatSetSizes(*A,m,n,m,n);CHKERRQ(ierr);
  ierr = MatSetType(*A,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
  ierr = MatSeqAIJSetPreallocation_SeqAIJ(*A,nz,(PetscInt*)nnz);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDestroy_SeqAIJCUSPARSE(Mat A)
{
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  if (A->factortype==MAT_FACTOR_NONE) {
    if (A->valid_GPU_matrix != PETSC_OFFLOAD_UNALLOCATED) {
      ierr = MatSeqAIJCUSPARSE_Destroy((Mat_SeqAIJCUSPARSE**)&A->spptr);CHKERRQ(ierr);
    }
  } else {
    ierr = MatSeqAIJCUSPARSETriFactors_Destroy((Mat_SeqAIJCUSPARSETriFactors**)&A->spptr);CHKERRQ(ierr);
  }
  ierr = MatDestroy_SeqAIJ(A);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDuplicate_SeqAIJCUSPARSE(Mat A,MatDuplicateOption cpvalues,Mat *B)
{
  PetscErrorCode ierr;
  Mat C;
  hipsparseStatus_t stat;
  hipsparseHandle_t handle=0;

  PetscFunctionBegin;
  ierr = MatDuplicate_SeqAIJ(A,cpvalues,B);CHKERRQ(ierr);
  C    = *B;
  ierr = PetscFree(C->defaultvectype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(VECCUDA,&C->defaultvectype);CHKERRQ(ierr);

  /* inject CUSPARSE-specific stuff */
  if (C->factortype==MAT_FACTOR_NONE) {
    /* you cannot check the inode.use flag here since the matrix was just created.
       now build a GPU matrix data structure */
    C->spptr = new Mat_SeqAIJCUSPARSE;
    ((Mat_SeqAIJCUSPARSE*)C->spptr)->mat          = 0;
    ((Mat_SeqAIJCUSPARSE*)C->spptr)->matTranspose = 0;
    ((Mat_SeqAIJCUSPARSE*)C->spptr)->workVector   = 0;
    ((Mat_SeqAIJCUSPARSE*)C->spptr)->format       = MAT_CUSPARSE_CSR;
    ((Mat_SeqAIJCUSPARSE*)C->spptr)->stream       = 0;
    ((Mat_SeqAIJCUSPARSE*)C->spptr)->handle       = 0;
    stat = hipsparseCreate(&handle);CHKERRCUDA(stat);
    ((Mat_SeqAIJCUSPARSE*)C->spptr)->handle       = handle;
    ((Mat_SeqAIJCUSPARSE*)C->spptr)->stream       = 0;
    ((Mat_SeqAIJCUSPARSE*)C->spptr)->nonzerostate = 0;
  } else {
    /* NEXT, set the pointers to the triangular factors */
    C->spptr = new Mat_SeqAIJCUSPARSETriFactors;
    ((Mat_SeqAIJCUSPARSETriFactors*)C->spptr)->loTriFactorPtr          = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)C->spptr)->upTriFactorPtr          = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)C->spptr)->loTriFactorPtrTranspose = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)C->spptr)->upTriFactorPtrTranspose = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)C->spptr)->rpermIndices            = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)C->spptr)->cpermIndices            = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)C->spptr)->workVector              = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)C->spptr)->handle                  = 0;
    stat = hipsparseCreate(&handle);CHKERRCUDA(stat);
    ((Mat_SeqAIJCUSPARSETriFactors*)C->spptr)->handle                  = handle;
    ((Mat_SeqAIJCUSPARSETriFactors*)C->spptr)->nnz                     = 0;
  }

  C->ops->assemblyend      = MatAssemblyEnd_SeqAIJCUSPARSE;
  C->ops->destroy          = MatDestroy_SeqAIJCUSPARSE;
  C->ops->setfromoptions   = MatSetFromOptions_SeqAIJCUSPARSE;
  C->ops->mult             = MatMult_SeqAIJCUSPARSE;
  C->ops->multadd          = MatMultAdd_SeqAIJCUSPARSE;
  C->ops->multtranspose    = MatMultTranspose_SeqAIJCUSPARSE;
  C->ops->multtransposeadd = MatMultTransposeAdd_SeqAIJCUSPARSE;
  C->ops->duplicate        = MatDuplicate_SeqAIJCUSPARSE;

  ierr = PetscObjectChangeTypeName((PetscObject)C,MATSEQAIJCUSPARSE);CHKERRQ(ierr);

  C->valid_GPU_matrix = PETSC_OFFLOAD_UNALLOCATED;

  ierr = PetscObjectComposeFunction((PetscObject)C, "MatCUSPARSESetFormat_C", MatCUSPARSESetFormat_SeqAIJCUSPARSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PETSC_EXTERN PetscErrorCode MatCreate_SeqAIJCUSPARSE(Mat B)
{
  PetscErrorCode ierr;
  hipsparseStatus_t stat;
  hipsparseHandle_t handle=0;

  PetscFunctionBegin;
  ierr = MatCreate_SeqAIJ(B);CHKERRQ(ierr);
  ierr = PetscFree(B->defaultvectype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(VECCUDA,&B->defaultvectype);CHKERRQ(ierr);

  if (B->factortype==MAT_FACTOR_NONE) {
    /* you cannot check the inode.use flag here since the matrix was just created.
       now build a GPU matrix data structure */
    B->spptr = new Mat_SeqAIJCUSPARSE;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->mat          = 0;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->matTranspose = 0;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->workVector   = 0;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->format       = MAT_CUSPARSE_CSR;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->stream       = 0;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->handle       = 0;
    stat = hipsparseCreate(&handle);CHKERRCUDA(stat);
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->handle       = handle;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->stream       = 0;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->nonzerostate = 0;
  } else {
    /* NEXT, set the pointers to the triangular factors */
    B->spptr = new Mat_SeqAIJCUSPARSETriFactors;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->loTriFactorPtr          = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->upTriFactorPtr          = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->loTriFactorPtrTranspose = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->upTriFactorPtrTranspose = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->rpermIndices            = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->cpermIndices            = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->workVector              = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->handle                  = 0;
    stat = hipsparseCreate(&handle);CHKERRCUDA(stat);
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->handle                  = handle;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->nnz                     = 0;
  }

  B->ops->assemblyend      = MatAssemblyEnd_SeqAIJCUSPARSE;
  B->ops->destroy          = MatDestroy_SeqAIJCUSPARSE;
  B->ops->setfromoptions   = MatSetFromOptions_SeqAIJCUSPARSE;
  B->ops->mult             = MatMult_SeqAIJCUSPARSE;
  B->ops->multadd          = MatMultAdd_SeqAIJCUSPARSE;
  B->ops->multtranspose    = MatMultTranspose_SeqAIJCUSPARSE;
  B->ops->multtransposeadd = MatMultTransposeAdd_SeqAIJCUSPARSE;
  B->ops->duplicate        = MatDuplicate_SeqAIJCUSPARSE;

  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSEQAIJCUSPARSE);CHKERRQ(ierr);

  B->valid_GPU_matrix = PETSC_OFFLOAD_UNALLOCATED;

  ierr = PetscObjectComposeFunction((PetscObject)B, "MatCUSPARSESetFormat_C", MatCUSPARSESetFormat_SeqAIJCUSPARSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*MC
   MATSEQAIJCUSPARSE - MATAIJCUSPARSE = "(seq)aijcusparse" - A matrix type to be used for sparse matrices.

   A matrix type type whose data resides on Nvidia GPUs. These matrices can be in either
   CSR, ELL, or Hybrid format. The ELL and HYB formats require CUDA 4.2 or later.
   All matrix calculations are performed on Nvidia GPUs using the CUSPARSE library.

   Options Database Keys:
+  -mat_type aijcusparse - sets the matrix type to "seqaijcusparse" during a call to MatSetFromOptions()
.  -mat_cusparse_storage_format csr - sets the storage format of matrices (for MatMult and factors in MatSolve) during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid).
.  -mat_cusparse_mult_storage_format csr - sets the storage format of matrices (for MatMult) during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid).

  Level: beginner

.seealso: MatCreateSeqAIJCUSPARSE(), MATAIJCUSPARSE, MatCreateAIJCUSPARSE(), MatCUSPARSESetFormat(), MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
M*/

PETSC_EXTERN PetscErrorCode MatGetFactor_seqaijcusparse_cusparse(Mat,MatFactorType,Mat*);


PETSC_EXTERN PetscErrorCode MatSolverTypeRegister_CUSPARSE(void)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSolverTypeRegister(MATSOLVERCUSPARSE,MATSEQAIJCUSPARSE,MAT_FACTOR_LU,MatGetFactor_seqaijcusparse_cusparse);CHKERRQ(ierr);
  ierr = MatSolverTypeRegister(MATSOLVERCUSPARSE,MATSEQAIJCUSPARSE,MAT_FACTOR_CHOLESKY,MatGetFactor_seqaijcusparse_cusparse);CHKERRQ(ierr);
  ierr = MatSolverTypeRegister(MATSOLVERCUSPARSE,MATSEQAIJCUSPARSE,MAT_FACTOR_ILU,MatGetFactor_seqaijcusparse_cusparse);CHKERRQ(ierr);
  ierr = MatSolverTypeRegister(MATSOLVERCUSPARSE,MATSEQAIJCUSPARSE,MAT_FACTOR_ICC,MatGetFactor_seqaijcusparse_cusparse);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


static PetscErrorCode MatSeqAIJCUSPARSE_Destroy(Mat_SeqAIJCUSPARSE **cusparsestruct)
{
  hipsparseStatus_t stat;
  hipsparseHandle_t handle;

  PetscFunctionBegin;
  if (*cusparsestruct) {
    MatSeqAIJCUSPARSEMultStruct_Destroy(&(*cusparsestruct)->mat,(*cusparsestruct)->format);
    MatSeqAIJCUSPARSEMultStruct_Destroy(&(*cusparsestruct)->matTranspose,(*cusparsestruct)->format);
    delete (*cusparsestruct)->workVector;
    if (handle = (*cusparsestruct)->handle) {
      stat = hipsparseDestroy(handle);CHKERRCUDA(stat);
    }
    delete *cusparsestruct;
    *cusparsestruct = 0;
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode CsrMatrix_Destroy(CsrMatrix **mat)
{
  PetscFunctionBegin;
  if (*mat) {
    delete (*mat)->values;
    delete (*mat)->column_indices;
    delete (*mat)->row_offsets;
    delete *mat;
    *mat = 0;
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEMultStruct_Destroy(Mat_SeqAIJCUSPARSETriFactorStruct **trifactor)
{
  hipsparseStatus_t stat;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  if (*trifactor) {
    if ((*trifactor)->descr) { stat = hipsparseDestroyMatDescr((*trifactor)->descr);CHKERRCUDA(stat); }
    if ((*trifactor)->solveInfo) { stat = cusparseDestroySolveAnalysisInfo((*trifactor)->solveInfo);CHKERRCUDA(stat); }
    ierr = CsrMatrix_Destroy(&(*trifactor)->csrMat);CHKERRQ(ierr);
    delete *trifactor;
    *trifactor = 0;
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEMultStruct_Destroy(Mat_SeqAIJCUSPARSEMultStruct **matstruct,MatCUSPARSEStorageFormat format)
{
  CsrMatrix        *mat;
  hipsparseStatus_t stat;
  hipError_t      err;

  PetscFunctionBegin;
  if (*matstruct) {
    if ((*matstruct)->mat) {
      if (format==MAT_CUSPARSE_ELL || format==MAT_CUSPARSE_HYB) {
        hipsparseHybMat_t hybMat = (hipsparseHybMat_t)(*matstruct)->mat;
        stat = hipsparseDestroyHybMat(hybMat);CHKERRCUDA(stat);
      } else {
        mat = (CsrMatrix*)(*matstruct)->mat;
        CsrMatrix_Destroy(&mat);
      }
    }
    if ((*matstruct)->descr) { stat = hipsparseDestroyMatDescr((*matstruct)->descr);CHKERRCUDA(stat); }
    delete (*matstruct)->cprowIndices;
    if ((*matstruct)->alpha)     { err=hipFree((*matstruct)->alpha);CHKERRCUDA(err); }
    if ((*matstruct)->beta_zero) { err=hipFree((*matstruct)->beta_zero);CHKERRCUDA(err); }
    if ((*matstruct)->beta_one)  { err=hipFree((*matstruct)->beta_one);CHKERRCUDA(err); }
    delete *matstruct;
    *matstruct = 0;
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSETriFactors_Destroy(Mat_SeqAIJCUSPARSETriFactors** trifactors)
{
  hipsparseHandle_t handle;
  hipsparseStatus_t stat;

  PetscFunctionBegin;
  if (*trifactors) {
    MatSeqAIJCUSPARSEMultStruct_Destroy(&(*trifactors)->loTriFactorPtr);
    MatSeqAIJCUSPARSEMultStruct_Destroy(&(*trifactors)->upTriFactorPtr);
    MatSeqAIJCUSPARSEMultStruct_Destroy(&(*trifactors)->loTriFactorPtrTranspose);
    MatSeqAIJCUSPARSEMultStruct_Destroy(&(*trifactors)->upTriFactorPtrTranspose);
    delete (*trifactors)->rpermIndices;
    delete (*trifactors)->cpermIndices;
    delete (*trifactors)->workVector;
    if (handle = (*trifactors)->handle) {
      stat = hipsparseDestroy(handle);CHKERRCUDA(stat);
    }
    delete *trifactors;
    *trifactors = 0;
  }
  PetscFunctionReturn(0);
}

